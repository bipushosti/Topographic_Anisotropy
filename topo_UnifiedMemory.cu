#include "hip/hip_runtime.h"




#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <ctype.h>
#include <getopt.h>
#include <string.h>

#include <stdio.h>
#include <cmath>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/device_vector.h>

using namespace std;

//Size of the GPU memory
#define GPU_MEMSIZE_GB		2

//For case in which XSIZE = 1201 and YSIZE = 801
#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280

//MAX_XSIZE_POSSIBLE is the maximum size of x or max number of columns if there is only one row
#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


//Always have even number of radius;and divisible by 10

#define	RADSTEP			1
#define ANGLESIZE		36	

#define PI 3.14

#define THREADS_PER_BLOCK	512

//#define FILENAME	"Annie_coastDEM.txt"
//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(float* data,float* angle,float* anisotropy,float* azimuth,float* variance,float* orientation,float* ortho,size_t XSIZE,size_t YSIZE,int RADIUS,int WINDOW_SIZE);
int Get_GPU_devices(void);
static void HandleError( hipError_t err,const char *file, int line );
//--------------------------------------------------------------------------------------------------------------------------//

//Current Usage:
//Global Memory: 773 MB


__global__ void getMatrix(float* data,float* angle,float* anisotropy,float* azimuth,float* variance,float* orientation,float* ortho,size_t XSIZE,size_t YSIZE,int RADIUS,int WINDOW_SIZE)
//__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,size_t XSIZE,size_t YSIZE)
{


//	Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int id = y * XSIZE + x;
//----------------------------------------------------------------------------------------------------------------------------//	

	if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
	else
	{

		//Actual computation
		int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
		int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j,k;
		//printf("Radius is: %d\n",RADIUS);
	
/*
		float variance[RADIUS];
		float orientation[RADIUS];
		float ortho[RADIUS];
*/		
		

		float value,sum_value,avg_value;
		float sum_valueOrtho,avg_valueOrtho;
		
	//	Initializing declared variables
		sum_value = 0;
		avg_value = 0;
		sum_valueOrtho = 0;
		avg_valueOrtho = 0;

	//	Iniitalize variance, ortho, and orientation arrays with max float value SGR changed i<100 to i<RADIUS

		
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			
			for(k = 0;k<RADIUS;k+=RADSTEP) {
	
				//Initializing to 0 so that the sum is zero everytime it starts
				sum_value = 0;
				sum_valueOrtho = 0;

				for(j=k;j<k+WINDOW_SIZE;j++){

					//Computation for angle of interest
					xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
					yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

					value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
					value = value * value;
				
					//One eighty angle computation
					xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
					yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
					valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
					valueOneEighty = valueOneEighty * valueOneEighty;

					sum_value = sum_value + value + valueOneEighty;
					avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

					//Computation for values on angle orthogonal to angle of interest
					xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
					yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
					valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
					valueOrtho1 = valueOrtho1 * valueOrtho1;

					//One eighty ortho angle computation
					xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
					yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

					valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
					valueOrtho2 = valueOrtho2 * valueOrtho2;

					sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
					avg_valueOrtho = sum_valueOrtho/(2*j+1);

					//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
					if(avg_value == 0) {
							if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
								avg_value = avg_valueOrtho;
							}
							else {
								avg_value = 1;
							}
					}

					if(avg_valueOrtho == 0) {
						avg_valueOrtho = 1;
					}
				
					//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
					if(avg_value < variance[id * RADIUS + j]) {
							variance[id * RADIUS + j] = avg_value;
							orientation[id * RADIUS + j] = angle[i];
							ortho[id * RADIUS + j] = avg_valueOrtho;		
					}	
				}
			}
		}
		for(j=0;j<RADIUS;j+=RADSTEP){	
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = (36+ortho[id * RADIUS + j])/(36+variance[id * RADIUS + j]);
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[id * RADIUS + j] * 180/PI;
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//

//--------------------------------------Handle Error()-----------------------------------------------------------//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
		cout << hipGetErrorString(err) << "in" << file << "at line" << line << "\n";
        exit( EXIT_FAILURE );
    }
}

//--------------------------------------Get_GPU_devices()-----------------------------------------------------------//
int Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));

	if(!prop.deviceOverlap){
		cout<< "Device does not handle overlaps so streams are not possible\n";
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		cout<<  DeviceCount <<"Devices Found\n";
	}else{
		cout<< "No devices found or error in reading the number of devices\n";
		return 0;
	}
	
	for(int i = 0;i<DeviceCount;i++){
		hipDeviceProp_t properties;
		HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
		cout<<"Device Number:"<< i << "\n";
		cout<<"  Device name: "<< properties.name;
		cout<<"  Device Global Memory size: "<< properties.totalGlobalMem/1000000 << "MB \n";
		cout<<"\n";

	}

	return DeviceCount;
}

//-------------------------------------------------------------------------------------------------------------//

int main(int argc,char* argv[])
{

	char FileName[20];
	char delimiterStr[10];
	char delimiter;
	int RADIUS;
	int WINDOW_SIZE;	

	//delimiter_string = "A";
	if(argc != 9){
		printf("\tArguments needed = 9; Provided = %d\n",argc);
                printf("Usage: ./Executable -i InputDataFileName -d Delimiter -r Radius -w WindowSize\n");
		printf("Exiting program\n");
		return 0;
	}


	int option;

	while ((option = getopt(argc, argv,"i:d:r:w:")) != -1) {
		switch (option) {
                        case 'i' : strcpy(FileName,optarg);
                            break;
                        case 'd' : strcpy(delimiterStr,optarg);
                            break;
                        case 'r' : RADIUS  = atoi(optarg);
                            break;
                        case 'w' : WINDOW_SIZE = atoi(optarg);
                            break;
                        default: printf("Usage: Executable -i InputDataFileName -d Delimiter -r Radius -w WindowSize\n");
                            exit(EXIT_FAILURE);
                }
         }


	//In the future use optarg
	if(strcmp(delimiterStr,"space")==0){
		delimiter = ' ';
	}
	else if(strcmp(delimiterStr,"Space")==0){
		delimiter = ' ';
	}
	else if(strcmp(delimiterStr,"tab")==0){
		delimiter = '\t';
	}
	else if(strcmp(delimiterStr,"Tab")==0){
		delimiter = '\t';
	}
	else{
		delimiter = delimiterStr[0];
	}
	
	printf("Delimiter: %c\n",delimiter);


	printf("Radius is %d\n",RADIUS);
	printf("Input file name is: %s\n",FileName);

//-------------------------------------------------------------------------------------//

	//Setting the output buffer to 500MB
	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);

	//File declarations and opening them
	FILE *datTxt1,*datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open inpcheck.txt file");
		return (-1);
	}
	

	datTxt1 = fopen(FileName,"r");	
	if(datTxt1 == NULL) {
		cout<< "Cannot open file:" << argv[1] <<  "\nCheck if file exists.\n";
		exit(1);
	}
	outputAnisotropy00 = fopen("outputDataAni_First.txt","w");
	outputAnisotropy09 = fopen("outputDataAni_Rad_div_10.txt","w");
	outputAnisotropy49 = fopen("outputDataAni_Rad_div_2.txt","w");
	outputAnisotropy99 = fopen("outputDataAni_Last.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi_First.txt","w");
	outputAzimuth09 = fopen("outputDataAzi_Rad_div_10.txt","w");
	outputAzimuth49 = fopen("outputDataAzi_Rad_div_2.txt","w");
	outputAzimuth99 = fopen("outputDataAzi_Last.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}


//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//
	size_t XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line)if(*max_line++ == ' ')++XSIZE;
	XSIZE+=1;
	
	


	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	fclose(datTxt1);
	cout<< "(XSIZE,YSIZE)::"<< "(" << XSIZE << "," << YSIZE << ")" << "\n";

	datTxt = fopen(FileName,"r");

	if(datTxt == NULL) {
		printf("Cannot open file: %s\nCheck if file exists\n",argv[1]);
		exit(1);
	}

//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//
	cout<< "(XSIZE,YSIZE)::"<< "(" << XSIZE << "," << YSIZE << ")" << "\n";
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
//	Allocating Managed Memory (Unified Memory)

//	dim3 gridSize(XSIZE ,(YSIZE+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,1);
//	dim3 blockSize(1,THREADS_PER_BLOCK,1);

	long int total_threads;

	float* data;
	float* anisotropy,*azimuth,*angle;
	float* variance,*orientation,*ortho;

	total_threads = THREADS_PER_BLOCK * ((XSIZE + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK) * YSIZE;

	HANDLE_ERROR(hipMallocManaged((void**)&angle,ANGLESIZE * sizeof(float)));
	HANDLE_ERROR(hipMallocManaged((void**)&data,XSIZE * YSIZE * sizeof(float)));
	HANDLE_ERROR(hipMallocManaged((void**)&anisotropy,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float)));
	HANDLE_ERROR(hipMallocManaged((void**)&azimuth,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float)));

	HANDLE_ERROR(hipMallocManaged((void**)&variance,total_threads * RADIUS * sizeof(float)));
	HANDLE_ERROR(hipMallocManaged((void**)&orientation,total_threads * RADIUS * sizeof(float)));
	HANDLE_ERROR(hipMallocManaged((void**)&ortho,total_threads * RADIUS * sizeof(float)));
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[XSIZE * 10 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	int Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 5 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	cout<< "Working1\n";
	while(fgets(line,XSIZE *10 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		cout << "Working2\n";
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d ",Value);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d\n",Value);
			}
		}
		
		j++;
	}	
	
	
//------------------------------------Matrix Declarations--------------------------------------------------------------------------------------------------------------//
//	float angle[ANGLESIZE];

	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
	}
	
	for(i=0;i<RADIUS * total_threads ;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
	}
//--------------------------------------CUDA-------------------------------------------------------------------------------------------------------------------------//

	
	hipError_t error;
	//error = hipSetDevice(Get_GPU_devices() -1);
	error = hipSetDevice(0);

	if(error == hipSuccess){
		 cout <<"success\n";
	}else{
		cout <<"unsuccessful\n";
	}
	
	//hipSetDevice(1);

	cout<< "Hello1\n";

	dim3 gridSize((XSIZE + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,YSIZE,1);
	dim3 blockSize(THREADS_PER_BLOCK,1,1);

	//dim3 gridSize(3,YSIZE,1);
	

	cout <<"Hello2\n";

	getMatrix<<<gridSize,blockSize>>>(data,angle,anisotropy,azimuth,variance,orientation,ortho,XSIZE,YSIZE,RADIUS,WINDOW_SIZE);

	error = hipDeviceSynchronize();
	if(error != hipSuccess){
		cout << "CUDA Device Synchronization Error:" << hipGetErrorString(error) << "\n";

    		// we can't recover from the error -- exit the program
    		return 0;
  	}

	error = hipGetLastError();

	if(error != hipSuccess){
		cout <<"CUDA Error:" << hipGetErrorString(error) << "\n";
    		// we can't recover from the error -- exit the program
	    	return 0;
  	}

	cout << "Hello3\n";
	cout << "Hello4\n";
	cout << "Hello5\n";

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
//			Writing to files


	for(j=0;j<YSIZE ;j++) {
		for(i=0;i<XSIZE ;i++) {
			if((j>(YSIZE - RADIUS - 1))||(j<(RADIUS))) continue;
			if((i>(XSIZE - RADIUS - 1))||(i<(RADIUS))) continue;

			if (i == (XSIZE  - RADIUS - 1)) {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\n");
				fprintf(outputAzimuth00,"\n");

				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP +RADIUS/10 -1]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAnisotropy09,"\n");
				fprintf(outputAzimuth09,"\n");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAnisotropy49,"\n");
				fprintf(outputAzimuth49,"\n");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS -1]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAnisotropy99,"\n");
				fprintf(outputAzimuth99,"\n");
			}
			else {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\t");
				fprintf(outputAzimuth00,"\t");
	
				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAnisotropy09,"\t");
				fprintf(outputAzimuth09,"\t");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);	
				fprintf(outputAnisotropy49,"\t");
				fprintf(outputAzimuth49,"\t");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAnisotropy99,"\t");
				fprintf(outputAzimuth99,"\t");	
			}					
		}
	}	


	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);
	
	hipFree(data);
	hipFree(angle);
	hipFree(azimuth);
	hipFree(anisotropy);
	hipFree(variance);
	hipFree(orientation);
	hipFree(ortho);
	
	
	//free(max_line);
//	free(anisotropy);
//	free(azimuth);

//	size_t free_byte ;

//	size_t total_byte ;
/*
	hipMemGetInfo( &free_byte, &total_byte );
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;
//	cout << "GPU memory usage: used = %f, free = %f MB, total = %f MB\n",used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
*/
	return 0;
}
