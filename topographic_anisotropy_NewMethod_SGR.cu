
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <limits.h>

#define XSIZE 	1201
#define YSIZE	801



#define RADIUS		100
#define	RADSTEP		1
#define ANGLESIZE	36	


#define PI 3.141592653589793

int main()
{
	FILE *datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	int data[YSIZE][XSIZE];

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}
	//1200 ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char line[1200 * 5 +2+1200];
	memset(line, '\0', sizeof(line));
	char *startPtr,*endPtr;
	
	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}

	outputAnisotropy00 = fopen("outputDataAni00.txt","w");
	outputAnisotropy09 = fopen("outputDataAni09.txt","w");
	outputAnisotropy49 = fopen("outputDataAni49.txt","w");
	outputAnisotropy99 = fopen("outputDataAni99.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi00.txt","w");
	outputAzimuth09 = fopen("outputDataAzi09.txt","w");
	outputAzimuth49 = fopen("outputDataAzi49.txt","w");
	outputAzimuth99 = fopen("outputDataAzi99.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}

	int i,j,Value;
	j = 0;
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	while(fgets(line,1200 *5 + 2 + 1200,datTxt)!=NULL) {	
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d ",Value);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d\n",Value);
			}
		}
		
		j++;
	}	
	
	//Fine
	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
	}
	
	//Initializing 3D matrix anisotropy
	float*** anisotropy;
	anisotropy = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		anisotropy[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			anisotropy[i][j] = (float*)malloc(RADIUS * sizeof(float));
		}
	}

	//Initializing 3D matrix anzimuth
	float*** azimuth;
	azimuth = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		azimuth[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			azimuth[i][j] = (float*)malloc(RADIUS * sizeof(float));
		}
	}

	//Actual computation
	int xrad,yrad,x,y,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
	int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2;
	float variance[100];
	float orientation[100];
	float ortho[100];
	float value,sum_value,avg_value;
	float sum_valueOrtho,avg_valueOrtho;
	sum_value = 0;
	avg_value = 0;
	sum_valueOrtho = 0;
	avg_valueOrtho = 0;

	

	//y = 0;
	for(y=0;y<YSIZE;y++) {
		for(x = 0;x<XSIZE;x++) {
		/*for(x = 0;x<XSIZE+1;x++) {
			if(x==XSIZE) {
				y++;	
				if(y==YSIZE){
					x = XSIZE;
					continue;
				}
				x=0;
				continue;
			
			}
			*/
			if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) continue;
			if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) continue;	

			for(i=0;i<100;i++){
				variance[i] = FLT_MAX;
				ortho[i] = FLT_MAX;
			}
			
			
			//Flipped
			for(i=0;i<ANGLESIZE;i++) {
				//Initializing to 0 so that the sum is zero everytime it starts
				sum_value = 0;
				sum_valueOrtho = 0;
				for(j = 0;j<RADIUS;j+=RADSTEP) {
		
					//Computation for angle of interest
					xrad = (int)round(cos(angle[i]) * (j+1) + x);	
					yrad = (int)round(sin(angle[i]) * (j+1) + y);	

					value = data[y][x] - data[yrad][xrad];
					value = value * value;
					
					//One eighty angle computation
					xradOneEighty = (int)round(cos(angle[i]+PI) * (j+1) + x);	
					yradOneEighty = (int)round(sin(angle[i]+PI) * (j+1) + y);	
					
					valueOneEighty = data[y][x] - data[yradOneEighty][xradOneEighty];
					valueOneEighty = valueOneEighty * valueOneEighty;

					sum_value = sum_value + value + valueOneEighty;
					avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

					//Computation for values on angle orthogonal to angle of interest
					xradOrtho1 = (int)round(cos(angle[i]+PI/2) * (j+1) + x);	
					yradOrtho1 = (int)round(sin(angle[i]+PI/2) * (j+1) + y);	
					
					valueOrtho1 = data[y][x] - data[yradOrtho1][xradOrtho1];
					valueOrtho1 = valueOrtho1 * valueOrtho1;

					//One eighty ortho angle computation
					xradOrtho2 = (int)round(cos(angle[i]+PI*3/2) * (j+1) + x);	
					yradOrtho2 = (int)round(sin(angle[i]+PI*3/2) * (j+1) + y);	

					valueOrtho2 = data[y][x] - data[yradOrtho2][xradOrtho2];
					valueOrtho2 = valueOrtho2 * valueOrtho2;

					sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
					avg_valueOrtho = sum_valueOrtho/(2*j+1);

					//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
					if(avg_value == 0) {
							if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
								avg_value = avg_valueOrtho;
							}
							else {
								avg_value = 1;
							}
					}

					if(avg_valueOrtho == 0) {
						avg_valueOrtho = 1;
					}
					
					//printf("1(%d,%d)	%f	%f\n",(j+1),(i+1),variance[j],avg_value);
					//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
					if(avg_value < variance[j]) {
						//	printf("2(%d)	%f	%f\n",j,variance[j],avg_value);
							variance[j] = avg_value;
							orientation[j] = angle[i];
							ortho[j] = avg_valueOrtho;		
					}	
				}
			}
			
			//variance, ortho, and orientation arrays should represent the orientation of the minimum variance for all scales. Take ratio of ortho to variance to get anisotropy and convert orientation to degrees.
			for(j=0;j<RADIUS;j+=RADSTEP){
				anisotropy[y][x][j] = ortho[j]/variance[j];
				azimuth[y][x][j] = orientation[j] * 180/PI ;
				
				//printf("%f	%f\n",variance[j],anisotropy[y][x][j]);	
			}

			
//			Writing to files
			

			if (x == (XSIZE - RADIUS - 1)) {
				fprintf(outputAnisotropy00,"%f",anisotropy[y][x][0]);
				fprintf(outputAzimuth00,"%f",azimuth[y][x][0]);
				fprintf(outputAnisotropy00,"\n");
				fprintf(outputAzimuth00,"\n");

				fprintf(outputAnisotropy09,"%f",anisotropy[y][x][9]);
				fprintf(outputAzimuth09,"%f",azimuth[y][x][9]);
				fprintf(outputAnisotropy09,"\n");
				fprintf(outputAzimuth09,"\n");

				fprintf(outputAnisotropy49,"%f",anisotropy[y][x][49]);
				fprintf(outputAzimuth49,"%f",azimuth[y][x][49]);
				fprintf(outputAnisotropy49,"\n");
				fprintf(outputAzimuth49,"\n");

				fprintf(outputAnisotropy99,"%f",anisotropy[y][x][99]);
				fprintf(outputAzimuth99,"%f",azimuth[y][x][99]);
				fprintf(outputAnisotropy99,"\n");
				fprintf(outputAzimuth99,"\n");
			}
			else {
				fprintf(outputAnisotropy00,"%f",anisotropy[y][x][0]);
				fprintf(outputAzimuth00,"%f",azimuth[y][x][0]);
				fprintf(outputAnisotropy00,"\t");
				fprintf(outputAzimuth00,"\t");
				
				fprintf(outputAnisotropy09,"%f",anisotropy[y][x][9]);
				fprintf(outputAzimuth09,"%f",azimuth[y][x][9]);
				fprintf(outputAnisotropy09,"\t");
				fprintf(outputAzimuth09,"\t");

				fprintf(outputAnisotropy49,"%f",anisotropy[y][x][49]);
				fprintf(outputAzimuth49,"%f",azimuth[y][x][49]);	
				fprintf(outputAnisotropy49,"\t");
				fprintf(outputAzimuth49,"\t");

				fprintf(outputAnisotropy99,"%f",anisotropy[y][x][99]);
				fprintf(outputAzimuth99,"%f",azimuth[y][x][99]);
				fprintf(outputAnisotropy99,"\t");
				fprintf(outputAzimuth99,"\t");
				
			}					
		}			
			
	}


		
	
	


	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);

	//Freeing 3D matrix anisotropy
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(anisotropy[i][j]);
		}
		free(anisotropy[i]);
	}
	free(anisotropy);

	//Freeing 3D matrix azimuth
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(azimuth[i][j]);
		}
		free(azimuth[i]);
	}
	free(azimuth);
	
	return 0;
}
