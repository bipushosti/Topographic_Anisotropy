#include "hip/hip_runtime.h"




//Input file: space delimited

#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//Size of the GPU memory
#define GPU_MEMSIZE_GB		2

//For case in which XSIZE = 1201 and YSIZE = 801
#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280

//MAX_XSIZE_POSSIBLE is the maximum size of x or max number of columns if there is only one row
#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//#define XSIZE 		1201
//#define YSIZE			801


//Always have even number of radius;and divisible by 10
#define RADIUS			100
#define	RADSTEP			1
#define ANGLESIZE		36	

#define PI 3.14

//#define FILENAME	"Annie_coastDEM.txt"
//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,size_t XSIZE,size_t YSIZE);
int Get_GPU_devices();
static void HandleError( hipError_t err,const char *file, int line );
//--------------------------------------------------------------------------------------------------------------------------//

//Current Usage:
//Global Memory: 773 MB


__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,size_t XSIZE,size_t YSIZE)
{
//	SGR I don't see where XSIZE or YSIZE are defined...
	
	//Actual computation
/*	int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
	int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

//	Hardwired to be at 100 Radius now. This needs to change!
	float variance[100];
	float orientation[100];
	float ortho[100];


	float value,sum_value,avg_value;
	float sum_valueOrtho,avg_valueOrtho;

//	Initializing declared variables
	sum_value = 0;
	avg_value = 0;
	sum_valueOrtho = 0;
	avg_valueOrtho = 0;
*/
	//for(i=0;i<ANGLESIZE;i++) {
		//angle[i] = i * 5 * PI/180;
		//printf("%d	::	%f\n",i,angle[i]);
	//	printf("Array Size: %d\n",sizeof(angle)/sizeof(float));
	//}
//	Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//----------------------------------------------------------------------------------------------------------------------------//	

	if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
	else
	{

		//Actual computation
		int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
		int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

	
		float variance[RADIUS];
		float orientation[RADIUS];
		float ortho[RADIUS];


		float value,sum_value,avg_value;
		float sum_valueOrtho,avg_valueOrtho;
		
	//	Initializing declared variables
		sum_value = 0;
		avg_value = 0;
		sum_valueOrtho = 0;
		avg_valueOrtho = 0;

	//	Iniitalize variance, ortho, and orientation arrays with max float value SGR changed i<100 to i<RADIUS
		for(i=0;i<RADIUS;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
		}
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			//Initializing to 0 so that the sum is zero everytime it starts
			sum_value = 0;
			sum_valueOrtho = 0;
			for(j = 0;j<RADIUS;j+=RADSTEP) {
	
				//Computation for angle of interest
				xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

				value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
				value = value * value;
				
				//One eighty angle computation
				xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
				yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
				valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
				valueOneEighty = valueOneEighty * valueOneEighty;

				sum_value = sum_value + value + valueOneEighty;
				avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

				//Computation for values on angle orthogonal to angle of interest
				xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
				yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
				valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
				valueOrtho1 = valueOrtho1 * valueOrtho1;

				//One eighty ortho angle computation
				xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
				yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

				valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
				valueOrtho2 = valueOrtho2 * valueOrtho2;

				sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
				avg_valueOrtho = sum_valueOrtho/(2*j+1);

				//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
				if(avg_value == 0) {
						if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
							avg_value = avg_valueOrtho;
						}
						else {
							avg_value = 1;
						}
				}

				if(avg_valueOrtho == 0) {
					avg_valueOrtho = 1;
				}
				
				//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
				if(avg_value < variance[j]) {
						variance[j] = avg_value;
						orientation[j] = angle[i];
						ortho[j] = avg_valueOrtho;		
				}	
			}
		}
		for(j=0;j<RADIUS;j+=RADSTEP){	
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = ortho[j]/variance[j];
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[j] * 180/PI;
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//

//--------------------------------------Handle Error()-----------------------------------------------------------//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

//--------------------------------------Get_GPU_devices()-----------------------------------------------------------//
int Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));

	if(!prop.deviceOverlap){
		printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		//printf("%d Devices Found\n",DeviceCount);
	}else{
		printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	for(int i = 0;i<DeviceCount;i++){
		hipDeviceProp_t properties;
		HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", properties.name);
		printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
		printf("\n");

	}

	return DeviceCount;
}

//-------------------------------------------------------------------------------------------------------------//

int main(int argc,char* argv[])
//int main()
{
	if(argc == 1){
		printf("Not enough arguments\n");
		return 0;
	}
		
	#undef RADIUS
	#define RADIUS atoi(argv[2])
	//Setting the output buffer to 500MB
	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);

	//File declarations and opening them
	FILE *datTxt1,*datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open inpcheck.txt file");
		return (-1);
	}
	

	datTxt1 = fopen(argv[1],"r");	
	//datTxt1 = fopen("Annie_coastDEM.txt","r");
	if(datTxt1 == NULL) {
		printf("Cannot open file: %s  \nCheck if file exists.\n",argv[1]);
		exit(1);
	}
	outputAnisotropy00 = fopen("outputDataAni_First.txt","w");
	outputAnisotropy09 = fopen("outputDataAni_Rad_div_10.txt","w");
	outputAnisotropy49 = fopen("outputDataAni_Rad_div_2.txt","w");
	outputAnisotropy99 = fopen("outputDataAni_Last.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi_First.txt","w");
	outputAzimuth09 = fopen("outputDataAzi_Rad_div_10.txt","w");
	outputAzimuth49 = fopen("outputDataAzi_Rad_div_2.txt","w");
	outputAzimuth99 = fopen("outputDataAzi_Last.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}


//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//

	size_t XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line)if(*max_line++ == ' ')++XSIZE;
	XSIZE+=1;
	
	


	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	fclose(datTxt1);
	printf("(XSIZE,YSIZE)::(%zd,%zd)\n",XSIZE,YSIZE);

	datTxt = fopen(argv[1],"r");
//	datTxt = fopen("Annie_coastDEM.txt","r");
	if(datTxt == NULL) {
		//printf("Cannot open file: %s\nCheck if file exists\n",argv[1]);
		exit(1);
	}
//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//

	printf("(XSIZE,YSIZE):(%zd,%zd)\n",XSIZE,YSIZE);
	//printf("Maximum size possible = %f\nTotal size of current data(XSIZE * YSIZE) = %zd\n",MAX_XSIZE_POSSIBLE,XSIZE * YSIZE);
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//

	int data[YSIZE * XSIZE];

	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[XSIZE * 10 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	int Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 5 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	printf("Working1\n");
	while(fgets(line,XSIZE *10 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		printf("Working2\n");
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d ",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d\n",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);
			}
		}
		
		j++;
	}	
	
	
//------------------------------------Matrix Declarations--------------------------------------------------------------------------------------------------------------//
	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
		//printf("%d	::	%f\n",i,angle[i]);
	}

	float* anisotropy;
	anisotropy = (float*)malloc(YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
	float *azimuth;
	azimuth = (float*)malloc(YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));

	//anisotropy[0][0][99] = 834;
	
	
//--------------------------------------CUDA-------------------------------------------------------------------------------------------------------------------------//

	
	hipError_t error;
	error = hipSetDevice(Get_GPU_devices() -1);

	if(error == hipSuccess){
		 printf("success\n");
	}else{
		printf("unsuccessful\n");
	}

	int *data_ptr;
	float *anisotropy_ptr,*azimuth_ptr,*angle_ptr;

	//hipSetDevice(1);
	hipMalloc((void**)&data_ptr,XSIZE * YSIZE * sizeof(int));
	hipMemcpy(data_ptr,data,XSIZE * YSIZE * sizeof(int),hipMemcpyHostToDevice);

	hipMalloc((void**)&angle_ptr,ANGLESIZE * sizeof(float));
	hipMemcpy(angle_ptr,angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice);
	
	hipMalloc((void**)&anisotropy_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
	hipMalloc((void**)&azimuth_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));


	printf("Hello1\n");

	//dim3 gridSize(3,YSIZE,1);
	dim3 gridSize((XSIZE + 512 - 1)/512,YSIZE,1);
	dim3 blockSize(512,1,1);

	printf("Hello2\n");

	getMatrix<<<gridSize,blockSize>>>(data_ptr,angle_ptr,anisotropy_ptr,azimuth_ptr,XSIZE,YSIZE);

	error = hipDeviceSynchronize();
	if(error != hipSuccess)
  	{
		printf("CUDA Device Synchronization Error: %s\n", hipGetErrorString(error));

    	// we can't recover from the error -- exit the program
    	return 0;
  	}
	error = hipGetLastError();
	if(error != hipSuccess)
  	{
		printf("CUDA Error: %s\n", hipGetErrorString(error));

    	// we can't recover from the error -- exit the program
    	return 0;
  	}

	printf("Hello3\n");
	
	hipMemcpy(anisotropy,anisotropy_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(azimuth,azimuth_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);
	

	printf("Hello4\n");

	hipFree(data_ptr);
	hipFree(angle_ptr);
	hipFree(azimuth_ptr);
	hipFree(anisotropy_ptr);
	printf("Hello5\n");

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
//			Writing to files


	for(j=0;j<YSIZE ;j++) {
		for(i=0;i<XSIZE ;i++) {
			if((j>(YSIZE - RADIUS - 1))||(j<(RADIUS))) continue;
			if((i>(XSIZE - RADIUS - 1))||(i<(RADIUS))) continue;

			if (i == (XSIZE  - RADIUS - 1)) {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\n");
				fprintf(outputAzimuth00,"\n");

				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP +RADIUS/10 -1]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAnisotropy09,"\n");
				fprintf(outputAzimuth09,"\n");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAnisotropy49,"\n");
				fprintf(outputAzimuth49,"\n");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS -1]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAnisotropy99,"\n");
				fprintf(outputAzimuth99,"\n");
			}
			else {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\t");
				fprintf(outputAzimuth00,"\t");
	
				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAnisotropy09,"\t");
				fprintf(outputAzimuth09,"\t");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);	
				fprintf(outputAnisotropy49,"\t");
				fprintf(outputAzimuth49,"\t");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAnisotropy99,"\t");
				fprintf(outputAzimuth99,"\t");	
			}					
		}
	}	


	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);
	

	
	//free(max_line);
	free(anisotropy);
	free(azimuth);

	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte );
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;
	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	return 0;
}
