#include "hip/hip_runtime.h"
//Input file: space delimited

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <limits.h>

#include <unistd.h>
#include <ctype.h>
#include <getopt.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
#include <assert.h>

#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "topoMultiGPU_Header.h"

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Size of the GPU memory
#define GPU_MEMSIZE_GB		6

//For case in which XSIZE = 1201 and YSIZE = 801
#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280

//MAX_XSIZE_POSSIBLE is the maximum size of x or max number of columns if there is only one row
#define MAX_XSIZE_POSSIBLE	7483647
//#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 


//#define XSIZE 		1201
//#define YSIZE			801


//Always have even number of radius;and divisible by 10
//#define RADIUS			100

#define	RADSTEP			1

using namespace std;

//---------------------------- Grid Layout ------------------------------------------------------------------------------------------------------//
/*

	|
	|
	|
Angles	|
	|
      	|
	------------------------------------------
		Radius (Divided by radiusDiv)

	Max the angles can be is 36;

*/	


//Changes
//March 6: 
//		Individual if(threadIdx.x < (dividedRadius * angleSize)) is replaced by one long condition (Tested + Works)
//		

//---------------------------Function and Global variable declarations--------------------------------------------------------------------------//
__global__ void getMatrix(const int* __restrict__ data,const float* __restrict__ angle,float* __restrict__ anisotropy,float* __restrict__ azimuth,long int XSIZE,long int YSIZE,int RADIUS,int angleSize,int radiusDiv);
__device__ float check_if_zero(float value_to_check,float functionArg1, float functionArg2);

//__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,long int XSIZE,long int YSIZE,int RADIUS,int angleSize,int radiusDiv);
int Get_GPU_devices();
static void HandleError( hipError_t err,const char *file, int line);
inline hipError_t checkCuda(hipError_t result);

//-----------------------------------------------Device Functions-----------------------------------------------------------//

//Function that returns avg_valueOrtho if avg_value ==0 && avg_valueOrtho <1 && avg_valueOrtho >0
//Or returns 1 otherwise
__device__ float calculate_averageValue(float avg_value,float avg_valueOrtho)
{

	float x1,x2,x3;
	//float f1,f2,f3;

	x1 = ceilf(avg_value/FLT_MAX);
	x2 = ceilf(floorf(avg_valueOrtho)/FLT_MAX);
	x3 = ceilf(avg_valueOrtho/FLT_MAX);	
/*	
	f3 = x3 * avg_valueOrtho + (1 - x3);
	f2 = x2 + (1 - x2) * f3;
	f1 = x1 * avg_value + (1 - x1) * f2;
*/

	return (x1 * avg_value + (1 - x1) * (x2 + (1 - x2) * (x3 * avg_valueOrtho + (1 - x3))));

}

//Function that returns 1 if averageValueOrtho == 0 and averageValueOrtho otherwise
__device__ float calculate_averageValueOrtho(float averageValueOrtho)
{
	return (ceil(averageValueOrtho/FLT_MAX) * averageValueOrtho + (1 - ceil(averageValueOrtho/FLT_MAX)) * 1);

}

//Function that emulates if(a<b){...} but without the if statement to avoid warp divergence
__device__ float condition_if_lessthan(float valueA,float valueB,float result_ifAIsLessThanB, float result_ifAIsNotLessThanB)
{

	return ceilf(floorf(valueA/valueB)/FLT_MAX) * result_ifAIsNotLessThanB + (1.0 -  ceilf(floorf(valueA/valueB)/FLT_MAX) ) * result_ifAIsLessThanB;


}


//--------------------------------------------------------------------------------------------------------------------------//

__global__ void getMatrix(const int* __restrict__ data,const float* __restrict__ angle,float* __restrict__ anisotropy,float* __restrict__ azimuth,long int XSIZE,long int YSIZE,int RADIUS,int angleSize,int radiusDiv)
//__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,long int XSIZE,long int YSIZE,int RADIUS,int angleSize,int radiusDiv)
{

	//Block Indices
	//int block_id =  blockIdx.y * gridDim.x + blockIdx.x;

	//Thread indices; Using only the x dimensions
	//int thread_id = block_id * blockDim.x + threadIdx.x;

	//The entire radius cannot be used to create shared memory; So a smaller radius size is used
	//When RADIUS = 100 and radiusDiv = 5, dividedRadius=20
	int dividedRadius = RADIUS/radiusDiv;

	//int thread_y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int thread_x = 0;
	int thread_y = 0;
	
	//Data indices
	int dataIdx_x = blockIdx.x + RADIUS;
	int dataIdx_y = blockIdx.y + RADIUS;
	


//----------------------------------------------------------------------------------------------------------------------------//	
	//Shared memory that holds avg_value and avg_valueOrtho
	extern __shared__ float averages[];

	//if(thread_y == 0 && thread_x == 0){printf("Inside the kernel\n");}


	int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
	int xradOrtho2,yradOrtho2,i;

	int sum_value,sum_valueOrtho;

	//int valueOrtho1,valueOrtho2;
	//float value;
	float avg_value,avg_valueOrtho;
	float avg_valueSum;


	//For 1 GPU
	if(threadIdx.x < (dividedRadius * angleSize)) //For radius=100, total threads per block=736, checks if less than 720 (idx starts at 0)
	{
//Needs review; Need to add CUDA float to int and back functions?
		thread_x = threadIdx.x - (int)((float)threadIdx.x / (float)dividedRadius) * dividedRadius; 
		thread_y = threadIdx.x / dividedRadius;

		

		sum_value = 0;
		sum_valueOrtho = 0;

		//Each thread in x-axis represents radiusDiv numbers
		for(i=0;i<radiusDiv;i++){


			//------------------------------------------------------------------------------------------------------------//	
			//Computation for angle of interest
			xrad = (int)lrintf(cosf(angle[thread_y]) * (thread_x * radiusDiv + i + 1) + dataIdx_x);	
			yrad = (int)lrintf(sinf(angle[thread_y]) * (thread_x * radiusDiv + i + 1) + dataIdx_y);	

//Works till here; xrad and yrad for this and the working versions are the same **


			//value = data[dataIdx_y * XSIZE + dataIdx_x]  - data[yrad * XSIZE + xrad];
			//value = value * value;
		
			//One eighty angle computation
			xradOneEighty = (int)lrintf(cosf(angle[thread_y]+3.14159) * (thread_x * radiusDiv + i + 1) + dataIdx_x);	
			yradOneEighty = (int)lrintf(sinf(angle[thread_y]+3.14159) * (thread_x * radiusDiv + i + 1) + dataIdx_y);	
		
			valueOneEighty = data[dataIdx_y * XSIZE + dataIdx_x] - data[yradOneEighty * XSIZE + xradOneEighty];
			valueOneEighty = valueOneEighty * valueOneEighty;

			//------------------------------------------------------------------------------------------------------------//
			//Computation for values on angle orthogonal to angle of interest
			xradOrtho1 = (int)lrintf(cosf(angle[thread_y]+3.14159/2) * (thread_x * radiusDiv + i + 1) + dataIdx_x);	
			yradOrtho1 = (int)lrintf(sinf(angle[thread_y]+3.14159/2) * (thread_x * radiusDiv + i + 1) + dataIdx_y);	
		
			//One eighty ortho angle computation
			xradOrtho2 = (int)lrintf(cosf(angle[thread_y]+3.14159*3/2) * (thread_x * radiusDiv + i + 1) + dataIdx_x);	
			yradOrtho2 = (int)lrintf(sinf(angle[thread_y]+3.14159*3/2) * (thread_x * radiusDiv + i + 1) + dataIdx_y);	

			//valueOrtho1 = data[dataIdx_y * XSIZE + dataIdx_x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
			//valueOrtho1 = valueOrtho1 * valueOrtho1;
			//valueOrtho2 = data[dataIdx_y * XSIZE + dataIdx_x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
			//valueOrtho2 = valueOrtho2 * valueOrtho2;

//Both parts of the sum value are same in this and the working version



//All rads and one eighty values work!
			//-------------------------------Getting the sum values-------------------------------------------------------//
			//sum_value = value + valueOneEighty;
			sum_value = data[dataIdx_y * XSIZE + dataIdx_x]  - data[yrad * XSIZE + xrad];

			sum_value = sum_value * sum_value;

//Same sum_values too!

			sum_value += valueOneEighty;	
//Sum_value is correct	

		
			//sum_valueOrtho = valueOrtho1 + valueOrtho2;
			sum_valueOrtho = data[dataIdx_y * XSIZE + dataIdx_x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
			sum_valueOrtho = sum_valueOrtho * sum_valueOrtho; //valueOrtho1
//valueOrtho1 Or sum_valueOrtho till is point is correct
						

			sum_valueOrtho += (data[dataIdx_y * XSIZE + dataIdx_x]  - data[yradOrtho2 * XSIZE + xradOrtho2])*(data[dataIdx_y * XSIZE + dataIdx_x]  - data[yradOrtho2 * XSIZE + xradOrtho2]);
			
//Correct till here; Problem with the averages?

			//-----Storing the sum values in the shared memory array-----------------------------------------------------//
			averages[thread_y * RADIUS + thread_x * radiusDiv + i] = sum_value;
//Correct sum_values going to the correct positions

			averages[RADIUS * angleSize + thread_y * RADIUS + thread_x * radiusDiv + i] = sum_valueOrtho;



			//averages[2 * RADIUS * angleSize + thread_y * RADIUS + thread_x* radiusDiv + i] = sum_value;

			//printf("X Y Value dataIdx_x dataIdx_y xrad yrad %d %d %f %d %d %f %f \n",thread_x,thread_y,sum_value,dataIdx_x,dataIdx_y,xrad,yrad);
			//printf("X Y EleValue %d %d %f\n",thread_x,thread_y,data[thread_y * RADIUS + thread_x]);
//			printf("thread_x thread_y xrad yrad %d %d %d %d\n",thread_x,thread_y,xrad,yrad);
	
		}
	
			
	//}
		__syncthreads();



	//Single thread averaging over the row or x dimension (Using 1 thread per row)---------------------------------------------------------------------------//
	//The block only contains threads in x dimension. Therefore, threadIdx.x is the thread index.	
	//if(threadIdx.x < (dividedRadius * angleSize)){

		//Only getting thr threads from 0 to angleSize (0 to 35; ANGLESIZE = 36 is hard coded)
		// Going through each Anglesize (y-direction) and getting the average of the row (x-direction)
		if(threadIdx.x < angleSize)
		{

			//--------Getting the avg_value and storing it in the shared mem array "averages"-------------------------------------//
			//Can't use an int to store a float
			//sum_value = 0;

			avg_valueSum = 0.0;

			//Loop from the start of the row to the end of the row which is RADIUS away from the start
			for(i = 0; i < RADIUS; i++){
				avg_valueSum += averages[threadIdx.x * RADIUS + i];	
				avg_value = avg_valueSum/(2*(i+1));
				averages[threadIdx.x * RADIUS + i] = avg_value;

		

			}

			//-------Getting the avg_valueOrtho and storing it in the shared mem array---------------------------------------------//	
			//Can't use int to store a float
			//sum_valueOrtho = 0;

			//Reusing variable
			//Using avg_valueSum to store sum of avg_valueOrtho
			avg_valueSum = 0.0;

			//Looping through the row
			for(i = 0; i < RADIUS; i++){
				avg_valueSum += averages[RADIUS * angleSize + threadIdx.x * RADIUS + i];	
				avg_valueOrtho = avg_valueSum/(2*i+1);
				averages[RADIUS * angleSize + threadIdx.x * RADIUS + i] = avg_valueOrtho;

			}
		}
			//Now the first matrix has the averaged values (avg_value);
			//And the second has the averaged Ortho values (avg_valueOrtho);	
		//}




		__syncthreads();
	
//Correct upto this point;
//Result from this is in float while that from the previous version is in ints. Therefore two results differ by less than an int (Ex: 374 vs 374.200012)
//Need to change the original code to make sure the results are stored in floats by using CUDA's conversion functions

//------------------>> Fixed till here; Ints were used to store floats










	//-Multi-thread averaging----------------------------------------------------------------------------------------------------//	
//*************
//NEEDS TO BE IMPLEMENTED ONCE THE SINGE THREAD AVERAGE PART IS COMPLETE AND ACCURATE
//*************



/*
	//Uses multiple threads to get the average

	if((thread_x > 0) && (threadIdx.x < (dividedRadius * angleSize)))
	{
		int offsetStart,offsetEnd;

		//Getting the avg_value and storing it in the shared mem array "averages"
		for(radIdx=0;radIdx<radiusDiv;radIdx++){

			sum_value = 0;
			
			offsetStart = thread_y * RADIUS + thread_x * radiusDiv + radIdx;
			offsetEnd = thread_y * RADIUS;
 
			for(i = offsetStart; i>= offsetEnd; i--)
			{ 
			//for(i=(thread_x * radiusDiv + radIdx); i>=0; i--){
				sum_value += averages[i];
			}

			avg_value = sum_value/(2*(thread_x * radiusDiv + radIdx + 1));
			averages[2 * RADIUS * angleSize + thread_y * RADIUS + thread_x * radiusDiv + radIdx] = avg_value;
	
		}
		//Now the last matrix has the averaged values (avg_value);
		//First has the individual sum values; 
		//And the second has the individual sum of Ortho values;

		__syncthreads();
	
		//------------------------------------------------------------------------------------------------------------//	
		//Getting the avg_valueOrtho and storing it in the shared mem array
		for(radIdx=0;radIdx<radiusDiv;radIdx++){
		
			sum_valueOrtho = 0;

			offsetStart = RADIUS * angleSize + thread_y * RADIUS + thread_x * radiusDiv + radIdx;
			offsetEnd = RADIUS * angleSize + thread_y * RADIUS;

			for(i = offsetStart; i>= offsetEnd; i++)
			//for(i=(thread_x * radiusDiv + radIdx); i>=0; i--)
			{
				sum_valueOrtho += averages[i];
			}

			avg_valueOrtho = sum_valueOrtho/(2*(thread_x * radiusDiv + radIdx + 1));
			averages[thread_y * RADIUS + thread_x * radiusDiv + radIdx] = avg_valueOrtho;

		}

		//Now the first matrix has averaged Ortho values (avg_valueOrtho); 
		//The last matrix still has the averaged values (avg_values);
		//And the second has the individual sum of Ortho values;

		__syncthreads();
	}
*/
	//if(blockIdx.x==0 && blockIdx.y ==0){ printf("threadIdx.x = %d\n",threadIdx.x);}



	//Error Checking-------------------------------------------------------------------------------------------------------------//	
	//if(threadIdx.x < (dividedRadius * angleSize))
	//{
		//thread_x = threadIdx.x - (int)((float)threadIdx.x / (float)dividedRadius) * dividedRadius; 
		//thread_y = threadIdx.x / dividedRadius;
		//----Error checking-----------------------------------------------------------------------------------------//
		for(i=0;i<radiusDiv;i++){
			//Getting averages and storing them in variables 

			avg_value = averages[thread_y * RADIUS + thread_x * radiusDiv + i];			
			avg_valueOrtho = averages[ RADIUS * angleSize + thread_y * RADIUS + thread_x * radiusDiv + i];
		

	//New method that removes the if statements completely
	//Not sure if it works yet
			avg_value = calculate_averageValue(avg_value,avg_valueOrtho);
			avg_valueOrtho = calculate_averageValueOrtho(avg_valueOrtho);

	
			//Storing the averages back into the shared memory array
			averages[thread_y * RADIUS + thread_x * radiusDiv + i] = avg_value;
			averages[RADIUS * angleSize + thread_y * RADIUS + thread_x * radiusDiv + i] = avg_valueOrtho;


			//printf("%d %d %f %f\n",thread_y,thread_x,avg_value,avg_valueOrtho);
		}
	
	//}
//Works till here!

		__syncthreads();

	}

	//--------------Transposing the matrix to get the smallest number from the columns-----------------------------------//
//*********************
//TRANSPOSE NOT NECESSARY IF USING ONE THREAD PER COLUMN
//*********************
/*



		int i,j,k;
		for(radIdx=0;radIdx<radiusDiv;radIdx++){
		    //for(int n = 0; n<N*M; n++) {
			//int i = n/N;
			//int j = n%N;
			//dst[n] = src[M*j + i];
		    //}
			i = (thread_y * RADIUS + thread_x * radiusDiv + radIdx)/angleSize;
			j = (thread_y * RADIUS + thread_x * radiusDiv + radIdx)%angleSize;
			k = RADIUS * j + i;
			//Transposing the matrix but fitting it into the same matrix; This is done to get min values from each RADIUS
			averages[RADIUS * angleSize + thread_y * RADIUS + thread_x * radiusDiv + radIdx] = averages[2 * RADIUS * angleSize + k];

		}
		__syncthreads();	

		//Now the second has the transposed last matrix such that all the values fit;
		//The first matrix has averaged Ortho values (avg_valueOrtho); 
		//The last matrix still has the averaged values (avg_values);
	
	}
*/
//------------------------------------------------------------------------------------------------------------//
	//Finding the minimum over the columns
	//if(threadIdx.x < (dividedRadius * angleSize))
	//{

			
	if(threadIdx.x < RADIUS)
	{

		avg_value = averages[threadIdx.x];

		avg_valueSum = angle[0];
		
		avg_valueOrtho = averages[RADIUS * angleSize + threadIdx.x];


		//In the non shared memory version this looks like:
		/*
		if(avg_value < variance[j]) {
			variance[j] = avg_value;
			orientation[j] = angle[i];
			ortho[j] = avg_valueOrtho;		
		}
		*/

		for(i=0;i<angleSize;i++)
		{			
			//printf("Radius Angle current_value conditionalValue %d %d %f %f\n",threadIdx.x,i,averages[i*RADIUS + threadIdx.x],condition_if_lessthan(averages[i*RADIUS+threadIdx.x],avg_value,averages[i*RADIUS + threadIdx.x],avg_value));				

			//Same condition for each case : if(avg_value < variance[j]);  Therefore the floorf, or the equation part does not change just the variables do
			avg_valueSum = condition_if_lessthan(averages[i*RADIUS+threadIdx.x],avg_value,angle[i],avg_valueSum);
			avg_valueOrtho = condition_if_lessthan(averages[i*RADIUS+threadIdx.x],avg_value,averages[RADIUS * angleSize + i * RADIUS + threadIdx.x],avg_valueOrtho);
			avg_value = condition_if_lessthan(averages[i*RADIUS+threadIdx.x],avg_value,averages[i*RADIUS + threadIdx.x],avg_value);


//avg_value same for this and the non-shared memory code 
//Needed to move the avg_value calculation below others in the code above
//printf("%d %d %f %f %f\n",threadIdx.x,i,avg_value,avg_valueSum,avg_valueOrtho);	
//**Works till here
			

			/*
			avg_value = floorf(averages[i*RADIUS+threadIdx.x]/avg_value) * avg_value + (1.0 -  floorf(averages[i*RADIUS+threadIdx.x]/avg_value) ) * averages[i*RADIUS + threadIdx.x];
			avg_valueSum = floorf(averages[i*RADIUS+threadIdx.x]/avg_value) * avg_valueSum +  (1.0 -  floorf(averages[i*RADIUS+threadIdx.x]/avg_value) ) * angle[i];
			avg_valueOrtho = floorf(averages[i*RADIUS +threadIdx.x]/avg_value) * avg_valueOrtho + (1.0 -  floorf(averages[i*RADIUS+threadIdx.x]/avg_value) ) * averages[RADIUS * angleSize + i * RADIUS + threadIdx.x];
			*/

				
		}
//Incorrect order of division; avg_value/avg_valueOrtho was put in instead of acg_valueOrtho/avg_value
		printf("%d %d %d %d %f %f %f\n",threadIdx.x,i,dataIdx_x,dataIdx_y,avg_value,avg_valueSum,avg_valueOrtho);	
		anisotropy[dataIdx_y * XSIZE * RADIUS + dataIdx_x * RADIUS + threadIdx.x] = avg_valueOrtho/avg_value; 
		azimuth[dataIdx_y * XSIZE * RADIUS + dataIdx_x * RADIUS + threadIdx.x] = avg_valueSum * 180/3.14159; 
		
		__syncthreads();

/*
		anisotropy[dataIdx_y * XSIZE * RADIUS + dataIdx_x * RADIUS + threadIdx.x] = avg_value/avg_valueOrtho; 
		azimuth[dataIdx_y * XSIZE * RADIUS + dataIdx_x * RADIUS + threadIdx.x] = avg_valueSum * 180/3.14159; 
*/

	//	printf("dataIdx_x dataIdx_y Radius Ani Azi %d %d %d %f %f\n",dataIdx_x,dataIdx_y,threadIdx.x,avg_value/avg_valueOrtho,avg_valueSum * 180/3.14159);
		
	}
	
/*

		if(threadIdx.x < RADIUS)
		{
			//Minimum value stored in avg_value to save on registers (local vars)
			avg_value = averages[threadIdx.x];
			//Orientation stored in sum_value to save on registers (local vars)
			sum_value = angle[0];
			//Ortho value stored in avg_valueOrtho to save on registers (local vars)
			avg_valueOrtho = averages[RADIUS * angleSize + threadIdx.x];


			for(i=1;i<angleSize;i++){

				if(averages[i * RADIUS + threadIdx.x] < avg_value){
					avg_value = averages[i*RADIUS + threadIdx.x];
					sum_value = angle[i];
					avg_valueOrtho = averages[RADIUS * angleSize + i * RADIUS + threadIdx.x];
				}
			}


		}

	//}

		__syncthreads();

	//if(threadIdx.x < (dividedRadius * angleSize))
	//{
		if(threadIdx.x < RADIUS)
		{
			anisotropy[dataIdx_y * XSIZE * RADIUS + dataIdx_x * RADIUS + threadIdx.x] = avg_value/avg_valueOrtho; 
			azimuth[dataIdx_y * XSIZE * RADIUS + dataIdx_x * RADIUS + threadIdx.x] = sum_value * 180/3.14159; 
		
		}
		
	}//End of the long if(threadIdx.x <(dividedRadius * angleSize) condition
*/
	//------------------------------------------------------------------------------------------------------------//

	return;		

 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//



//--------------------------------------Handle Error()-----------------------------------------------------------//

static void HandleError( hipError_t err,const char *file, int line) 
{
	if (err != hipSuccess) 
	{
	        fprintf( stderr,"%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit(err);
    	}
}


//--------------------------------------Get_GPU_devices()-----------------------------------------------------------//

int Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));

	if(!prop.deviceOverlap){
		printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		printf("%d Devices Found\n",DeviceCount);
	}else{
		printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	for(int i = 0;i<DeviceCount;i++){
		hipDeviceProp_t properties;
		HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", properties.name);
		printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
		printf("\n");

	}

	return DeviceCount;
}



//#################################################################################################################################//
//#################################################################################################################################//
//#################################################################################################################################//

//--------------------------------------Main()-----------------------------------------------------------//
int main(int argc,char* argv[])
//int main()
{

	
	char FileName[20];
	char delimiterStr[10];
	char delimiter;
	int RADIUS;
	int WINDOW_SIZE;	
	//Size of angle array
	int ANGLESIZE = 36;	
	
	if(argc != 9){
		printf("\tArguments needed = 9; Provided = %d\n",argc);
                printf("Usage: ./Executable -i InputDataFileName -d Delimiter -r Radius -w WindowSize\n");
		printf("Exiting program\n");
		return 0;
	}


	int option;

	while ((option = getopt(argc, argv,"i:d:r:w:")) != -1) {
		switch (option) {
                        case 'i' : strcpy(FileName,optarg);
                            break;
                        case 'd' : strcpy(delimiterStr,optarg);
                            break;
                        case 'r' : RADIUS  = atoi(optarg);
                            break;
                        case 'w' : WINDOW_SIZE = atoi(optarg);
                            break;
                        default: printf("Usage: Executable -i InputDataFileName -d Delimiter -r Radius -w WindowSize\n");
                            exit(EXIT_FAILURE);
                }
         }


	//In the future use optarg
	if(strcmp(delimiterStr,"space")==0){
		delimiter = ' ';
	}
	else if(strcmp(delimiterStr,"Space")==0){
		delimiter = ' ';
	}
	else if(strcmp(delimiterStr,"tab")==0){
		delimiter = '\t';
	}
	else if(strcmp(delimiterStr,"Tab")==0){
		delimiter = '\t';
	}
	else{
		delimiter = delimiterStr[0];
	}
	
	printf("Delimiter: %c\n",delimiter);
	//return 0;

	//#undef RADIUS
	//#define RADIUS atoi(argv[3])

	//RADIUS = tmp;
	printf("Radius is %d\n",RADIUS);
	printf("AngleSize is %d\n",ANGLESIZE);
	printf("Input file name is: %s\n",FileName);


	if(RADIUS > 100){
		printf("Radius value cannot exceed 100\nExiting\n");
		return (-1);
	}
//-------------------------------------------------------------------------------------//	
	//File declarations and opening them
	FILE *datTxt1,*datTxt;
	FILE *outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy24,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth24,*outputAzimuth49,*outputAzimuth99; 
	
	FILE *outputAnisotropy04,*outputAzimuth04;
	FILE * inpCheck;


	datTxt1 = fopen(FileName,"r");	
	inpCheck = fopen("inpCheck.txt","w");

	if(inpCheck == NULL) {
		perror("Cannot open inpcheck.txt file");
		return (-1);
	}
		
	if(datTxt1 == NULL) {
		printf("Cannot open file: %s  \nCheck if file exists.\n",argv[1]);
		exit(1);
	}

//-------------------------------------------------------------------------------------//
//				Setting Up Output Filenames
//-------------------------------------------------------------------------------------//

	char AniFirst[80],AniFive[80],AniTen[80],AniTwentyFive[80],AniFifty[80],AniLast[80];
	char AziFirst[80],AziFive[80],AziTen[80],AziTwentyFive[80],AziFifty[80],AziLast[80];
	
	strcpy(AniFirst,"Out_Ani_First_");
	strcpy(AniFive,"Out_Ani_Five_");
	strcpy(AniTen,"Out_Ani_Ten_");
	strcpy(AniTwentyFive,"Out_Ani_TwentyFive_");
	strcpy(AniFifty,"Out_Ani_Fifty_");
	strcpy(AniLast,"Out_Ani_Last_");

	strcat(AniFirst,FileName);
	strcat(AniFive,FileName);
	strcat(AniTen,FileName);
	strcat(AniTwentyFive,FileName);
	strcat(AniFifty,FileName);
	strcat(AniLast,FileName);

	strcpy(AziFirst,"Out_Azi_First_");
	strcpy(AziFive,"Out_Azi_Five_");
	strcpy(AziTen,"Out_Azi_Ten_");
	strcpy(AziTwentyFive,"Out_Azi_TwentyFive_");
	strcpy(AziFifty,"Out_Azi_Fifty_");
	strcpy(AziLast,"Out_Azi_Last_");

	strcat(AziFirst,FileName);
	strcat(AziFive,FileName);
	strcat(AziTen,FileName);
	strcat(AziTwentyFive,FileName);
	strcat(AziFifty,FileName);
	strcat(AziLast,FileName);

	printf("Ani First is %s\n",AniFirst);
//-------------------------------------------------------------------------------------//


	outputAnisotropy00 = fopen(AniFirst,"a");
	outputAnisotropy04 = fopen(AniFive,"a");
	outputAnisotropy09 = fopen(AniTen,"a");
	outputAnisotropy24 = fopen(AniTwentyFive,"a");
	outputAnisotropy49 = fopen(AniFifty,"a");
	outputAnisotropy99 = fopen(AniLast,"a");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen(AziFirst,"a");
	outputAzimuth04 = fopen(AziFive,"a");
	outputAzimuth09 = fopen(AziTen,"a");
	outputAzimuth24 = fopen(AziTwentyFive,"a");
	outputAzimuth49 = fopen(AziFifty,"a");
	outputAzimuth99 = fopen(AziLast,"a");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}

//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//

	long int XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	long int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line !='\0'){
		if(*max_line == delimiter){
			XSIZE++;
		}
		max_line++;
	}
	
	XSIZE+=1;
	
	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	printf("(XSIZE,YSIZE)::(%ld,%ld)\n",XSIZE,YSIZE);

	datTxt = fopen(FileName,"r");
	if(datTxt == NULL) {
		printf("Cannot open file: %s\nCheck if file exists\n",argv[1]);
		exit(1);
	}
//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//

	printf("(XSIZE,YSIZE):(%ld,%ld)\n",XSIZE,YSIZE);
	//printf("Maximum size possible = %f\nTotal size of current data(XSIZE * YSIZE) = %zd\n",MAX_XSIZE_POSSIBLE,XSIZE * YSIZE);
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//


	float* data;

	data = (float*)malloc(YSIZE * XSIZE * sizeof(float));

	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[XSIZE * 10 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	float Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 7 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	printf("Reading the data file.\n");
	while(fgets(line,XSIZE *10 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		//printf("Working2\n");
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,delimiter);
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atof(tempVal);
				*(data + j * XSIZE + i) = Value;
				fprintf(inpCheck,"%f ",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);
				//printf("Column %d\n",i);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atof(tempVal);
				*(data + j * XSIZE + i) = Value;
				fprintf(inpCheck,"%f\n",Value);
					
			//	printf("(j,i)::(%d,%d)\n",j,i);
		
			}
		}
		
		j++;
	}
	printf("Closing the inputdata text files. \n");
	fclose(datTxt);
	fclose(datTxt1);
	fclose(inpCheck);

	printf("Done data[%zd][%zd] = %f\n",j-1,i-1,*(data + (j-1) * XSIZE + (i-1)));	
	printf("Working File IO\n");


	HANDLE_ERROR(hipDeviceSetLimit(hipLimitPrintfFifoSize, 10000000));
//-----------Getting the number of devices and their sizes------------------------------------------------//
        const int DeviceCount = Get_GPU_devices();


//------------------Initializing the structures that will hold GPU data-----------------------------------//
	GPU_struct GPU_values[DeviceCount];
	//HANDLE_ERROR(hipHostMalloc((void**)&GPU_values,DeviceCount*sizeof GPU_values));
//--------------------------------------------------------------------------------------------------------//

	//XSIZE  = number of total columns
	//YSIZE = number of total rows

	//Variable that holds number of rows for each GPU 
	int tmpSize = 0;
	//offset holds either 2*RADIUS or RADIUS depending on the part of data
	int offset = 0;
	//sum of the total positions of the rows for each GPU in each iteration
	int pos = 0;
	//Actual position inside the data matrix
	size_t data_position = 0;

	tmpSize = YSIZE/DeviceCount;
	printf("Each GPU gets %d rows\n",tmpSize);

	//Iterating through the available devices upto the second last
	for(i = 0;i<DeviceCount - 1;i++){

		printf("\n######################## Device %d #############################\n",i);

		
		GPU_values[i].NumCols = XSIZE;	

		//The last device is taken care after getting the remaining rows
		if(i == 0){
			GPU_values[i].NumRows = tmpSize + RADIUS;
			GPU_values[i].size = (GPU_values[i].NumRows + RADIUS ) * XSIZE;
			printf("Number of rows are: %ld\n",GPU_values[i].NumRows);
		//Sections in between
			printf("i is: %ld\n",i);
		}else{
			GPU_values[i].NumRows = tmpSize + 2 * RADIUS;
			GPU_values[i].size = (GPU_values[i].NumRows + 2*RADIUS) * XSIZE;
			printf("Number of rows are: %ld\n",GPU_values[i].size/XSIZE);
			//offset = RADIUS * -1;
		}
		printf("Size is: (GPU_values[%zd].NumRows + RADIUS) * XSIZE *sizeof(int) = (%ld + %d )*%ld *%ld =  %ld\n",i,GPU_values[i].NumRows,RADIUS,XSIZE,sizeof(float),GPU_values[i].size*sizeof(float));	
	}

	//---------------------Allocating number of rows to the last device--------------------------------//
	printf("\n########################Device %d ############################\n",DeviceCount -1);

	//Store the remaining rows in the last GPU
	GPU_values[DeviceCount - 1].NumRows = YSIZE - (tmpSize * (DeviceCount - 1)) + RADIUS;
	GPU_values[i].NumCols = XSIZE;	
	GPU_values[DeviceCount - 1].size = (GPU_values[DeviceCount - 1].NumRows + RADIUS) * XSIZE;

//--------------------If only a single GPU was found--------------------------------------------//
	if(DeviceCount == 1){
		GPU_values[0].NumRows = YSIZE;
		GPU_values[0].NumCols = XSIZE;
		GPU_values[0].size = YSIZE * XSIZE;

		printf("NumRows: %d, NumCols: %d\n",GPU_values[0].NumRows,GPU_values[0].NumCols);
	}

	//----------------------------------------------------------------------------------------------//
	printf("Number of rows are: %ld\n",GPU_values[DeviceCount - 1].NumRows);
	printf("i is: %ld\n",DeviceCount - 1);

	//----------------------------------------------------------------------------------------------//


	int numSegments = 1;
	
	for(i = 0;i<DeviceCount;i++){

		if((i==0) || (i==DeviceCount-1)){
			numSegments = 1;
		}else{
			numSegments = 2;
		}


		printf("\n########################Device %d #############################\n",i);
		printf("Radius is %d\n",RADIUS);
		//-----------------Matrix Allocations----------------------------//
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipStreamCreate(&GPU_values[i].stream));
		HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, (size_t)(GPU_values[i].size *sizeof(int) + ANGLESIZE * sizeof(float) + 8*GPU_values[i].size * RADIUS/RADSTEP * sizeof(float))));

		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_data,GPU_values[i].size * sizeof(int)));	
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_angle,ANGLESIZE * sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));

		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_data,GPU_values[i].size * sizeof(int)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_angle,ANGLESIZE * sizeof(float)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));

		//---------------Initialization of data arrays for each GPU---------// 
		//Populating the angle array
		for(j=0;j<ANGLESIZE;j++) {
			GPU_values[i].h_angle[j] = j * 5 * 3.14159/180;	
		}

		data_position = (pos + offset ) * XSIZE;

		printf("pos = %d,data_position after sub Index = %zd\n",pos,data_position);

		//Initializing the data arrays in each of the gpu with portions of the main data
		for(j=0;j<GPU_values[i].size;j++){
			GPU_values[i].h_data[j] = *(data + data_position+j);
		}
		
		printf("Data array assigned \n");
		offset = RADIUS * -1;
		pos+=GPU_values[i].NumRows-numSegments*RADIUS;	
	}

	//Shared memory only holds the anisotropy data for each point in the (radius,anglesize) grid
	size_t SharedMemSize = 2 * RADIUS * ANGLESIZE * sizeof(float);
	int threadsPBlock_X;
	int threadsPBlock_Y; 
	//Radius is divided by radiusDiv to make sure number of threads per block is less than max (1024)
	int radiusDiv;
	int tempSize;

//***NOTE***
	//Lowest number of threads per block that can be if only changing the X axis is : (32,64,1)
	//And 32 * 64 = 2048 which is greater than 1024!!
	//Both the RADIUS and the ANGLESIZE has to change!
	//BUT, does the total number of threads have to be multiple of one warp size (32) or 
	//	threads in each dimension have to be a multiple of one warp size (32)



//	(RADIUS % 32) == 0?:threadsPBlock_X = RADIUS:threadsPBlock_X = 32 * (RADIUS/32 + 1);
//	(ANGLESIZE % 32) == 0?:threadsPBlock_Y = ANGLESIZE;threadsPBlock_Y = 32 * (ANGLESIZE/32 + 1);


	//Creating a one dimensional thread block	
	threadsPBlock_Y = 1;

	//Getting the maximum possible threads per block as it cannot exceed 1024
	if( RADIUS * ANGLESIZE > 1024){

		for(radiusDiv=5; radiusDiv<10; radiusDiv++){

			if(RADIUS % radiusDiv == 0) {

				tempSize = RADIUS/radiusDiv * ANGLESIZE;
				if((tempSize % 32) != 0){
					tempSize = 32 * (tempSize/32 + 1);
				}

				if(tempSize < 1024){
					//threadsPBlock_X = RADIUS/radiusDiv;
					threadsPBlock_X = tempSize;
					break;
				}			
			}	
		}
	}
	//If the total threads per block is less than 1024
	else{
		//Since the total threads is less than 1024 there is no need to divide the RADIUS		
		radiusDiv = 1;
		threadsPBlock_X = RADIUS * ANGLESIZE;

		if(threadsPBlock_X % 32 != 0){
			threadsPBlock_X  = 32 * (threadsPBlock_X/32 + 1);
		}				
	}


	printf("radiusDiv is: %d\n",radiusDiv);
	printf("Total Threads per block is: %d \n",threadsPBlock_X);
	printf("SharedMemSize: %ld, threadsPBlock_X: %d, threadsPBlock_Y: %d\n",SharedMemSize,threadsPBlock_X,threadsPBlock_Y);
	printf("GridX: %d, GridY: %d\n",GPU_values[0].NumCols - 2* RADIUS,GPU_values[0].NumRows - 2* RADIUS);

	
	for(i=0;i<DeviceCount;i++){
		printf("\n########################Device %d #############################\n",i);
		HANDLE_ERROR(hipSetDevice(i));

		//-----------------Sending data to GPU----------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_data,GPU_values[i].h_data,GPU_values[i].size * sizeof(int),hipMemcpyHostToDevice,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_angle,GPU_values[i].h_angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice,GPU_values[i].stream));

		//----------------Kernel Variables---------------------//
		if((i==0) || (i==DeviceCount-1)){
			numSegments = 1;
		}else{
			numSegments = 2;

		}
		
		//For 1 GPU
		dim3 gridSize(GPU_values[i].NumCols - 2* RADIUS,GPU_values[i].NumRows - 2*RADIUS,1);
		dim3 blockSize(threadsPBlock_X,threadsPBlock_Y,1);

		//printf("GridSize(X,Y) = (%ld,%ld)\n",(GPU_values[i].NumCols + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,(GPU_values[i].NumRows));

		//----------------Launching the Kernel---------------------//
		printf("Kernel Launch # : %d\n",i);
		getMatrix<<<gridSize,blockSize,SharedMemSize,GPU_values[i].stream>>>(GPU_values[i].d_data,GPU_values[i].d_angle,GPU_values[i].d_anisotropy,GPU_values[i].d_azimuth,GPU_values[i].NumCols,GPU_values[i].NumRows,RADIUS,ANGLESIZE,radiusDiv);
		//HANDLE_ERROR(hipPeekAtLastError());

		//HANDLE_ERROR(hipDeviceSynchronize());
		//getLastCudaError("Kernel failed \n");

	}

	for(i=0;i<DeviceCount;i++){

		HANDLE_ERROR(hipDeviceSynchronize());
		//---------------Getting data back------------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_anisotropy,GPU_values[i].d_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_azimuth,GPU_values[i].d_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost,GPU_values[i].stream));

		printf("Device # %ld\n",i);
	}

	printf("\n");
	int z;
	//To offset the data by radius so that the read starts in the correct segment
	int offsetRadius = 0;

	for(z = 0;z<DeviceCount;z++){

		printf("\n###############################################################\n",z);
		printf("\n########################Device %d #############################\n",z);
		HANDLE_ERROR(hipSetDevice(z));
		HANDLE_ERROR(hipStreamSynchronize(GPU_values[z].stream));

		printf("Rows: %ld,Cols: %ld\n",GPU_values[z].NumRows,GPU_values[z].NumCols);
		printf("Radius is: %d\n",RADIUS);

		if((z==0) || z==(DeviceCount-1)){
			offsetRadius = 0;
		}else{
			offsetRadius = 1;
		}

		for(j=0;j<GPU_values[z].NumRows ;j++) {

			for(i=0;i<GPU_values[z].NumCols ;i++) {

				if((j>(GPU_values[z].NumRows - RADIUS - 1))||(j<(RADIUS))) continue;
				if((i>(GPU_values[z].NumCols - RADIUS - 1))||(i<(RADIUS))) continue;

				//printf("Col:%ld,Row: %ld\n",i,j);
				//If last element in the row
				if (i == (GPU_values[z].NumCols  - RADIUS - 1)) {
					fprintf(outputAnisotropy00,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAzimuth00,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAnisotropy00,"\n");
					fprintf(outputAzimuth00,"\n");

					fprintf(outputAnisotropy04,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAzimuth04,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAnisotropy04,"\n");
					fprintf(outputAzimuth04,"\n");

					fprintf(outputAnisotropy09,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 10]);
					fprintf(outputAzimuth09,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 10]);
					fprintf(outputAnisotropy09,"\n");
					fprintf(outputAzimuth09,"\n");

					fprintf(outputAnisotropy24,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAzimuth24,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAnisotropy24,"\n");
					fprintf(outputAzimuth24,"\n");

					fprintf(outputAnisotropy49,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAzimuth49,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAnisotropy49,"\n");
					fprintf(outputAzimuth49,"\n");

					fprintf(outputAnisotropy99,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAzimuth99,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAnisotropy99,"\n");
					fprintf(outputAzimuth99,"\n");

					
				}
				else {
					fprintf(outputAnisotropy00,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAzimuth00,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAnisotropy00,"\t");
					fprintf(outputAzimuth00,"\t");


					fprintf(outputAnisotropy04,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAzimuth04,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAnisotropy04,"\t");
					fprintf(outputAzimuth04,"\t");

					fprintf(outputAnisotropy09,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 10]);
					fprintf(outputAzimuth09,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 10]);
					fprintf(outputAnisotropy09,"\t");
					fprintf(outputAzimuth09,"\t");
	

					fprintf(outputAnisotropy24,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAzimuth24,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAnisotropy24,"\t");
					fprintf(outputAzimuth24,"\t");

					fprintf(outputAnisotropy49,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAzimuth49,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAnisotropy49,"\t");
					fprintf(outputAzimuth49,"\t");

					fprintf(outputAnisotropy99,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAzimuth99,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAnisotropy99,"\t");
					fprintf(outputAzimuth99,"\t");
	
				}
					
			}
		}
	}	


	for(i=0;i<DeviceCount;i++){
		//------------------------------------Freeing data---------------------------------------//
		HANDLE_ERROR(hipSetDevice(i));	
		HANDLE_ERROR(hipStreamSynchronize(GPU_values[i].stream));
		HANDLE_ERROR(hipDeviceSynchronize());

		HANDLE_ERROR(hipFree(GPU_values[i].d_anisotropy));
		HANDLE_ERROR(hipFree(GPU_values[i].d_azimuth));
		HANDLE_ERROR(hipFree(GPU_values[i].d_data));
		HANDLE_ERROR(hipFree(GPU_values[i].d_angle));

		HANDLE_ERROR(hipHostFree(GPU_values[i].h_data));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_angle));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_anisotropy));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_azimuth));

		HANDLE_ERROR(hipStreamDestroy(GPU_values[i].stream));

		hipDeviceReset();
			
	}

	/*for(i=0;i<DeviceCount;i++){
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_data));
		hipDeviceReset();
	}*/

	
	
	free(data);

	fclose(outputAnisotropy00);
	fclose(outputAnisotropy04);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy24);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth04);
	fclose(outputAzimuth09);
	fclose(outputAzimuth24);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);

	return 0;
}
		
		
