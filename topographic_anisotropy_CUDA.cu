


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

#define XSIZE 	1201
#define YSIZE	801


#define RADIUS		100
#define	RADSTEP		1
#define ANGLESIZE	72	


#define PI 3.141592653589793

int main()
{
	FILE *datTxt;
	int data[YSIZE][XSIZE];

	//1200 ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char line[1200 * 5 +2+1200];
	memset(line, '\0', sizeof(line));
	char *startPtr,*endPtr;
	
	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}

	int i,j,Value;
	j = 0;
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	while(fgets(line,1200 *5 + 2 + 1200,datTxt)!=NULL) {
	
		startPtr = line;
		//Skipping the first space
		startPtr+=1;
		//printf("%s",startPtr);

		
		for(i=0;i<XSIZE;i++) {
			endPtr = strchr(startPtr,' ');
			if(endPtr != NULL) {	
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j][i] = Value;
			}	
			endPtr = endPtr + 1;
			startPtr = endPtr;	
		}
		j++;
	}	


	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
		//printf("%d	::	%f\n",i,angle[i]);
	}
	

	//Initializing 2D cmatrix
	double** cmatrix;
	cmatrix = (double**)malloc(ANGLESIZE * sizeof(double*));
	for(i=0;i<ANGLESIZE;i++) {
		cmatrix[i] = (double*)malloc(RADIUS/RADSTEP *sizeof(double));
	}

	//Initializing cor
	double** cor;
	cor = (double**)malloc(ANGLESIZE * sizeof(double*));
	for(i=0;i<ANGLESIZE;i++) {
		cor[i] = (double*)malloc(RADIUS/RADSTEP *sizeof(double));
	}

	//Initializing cor_bi
	double** cor_bi;
	cor_bi = (double**)malloc(ANGLESIZE/2 * sizeof(double*));
	for(i=0;i<ANGLESIZE/2;i++) {
		cor_bi[i] = (double*)malloc(RADIUS/RADSTEP *sizeof(double));
	}

	//Initializing 3D matrix anisotropy
	double*** anisotropy;
	anisotropy = (double***)malloc(YSIZE * sizeof(double**));
	for(i = 0;i<YSIZE;i++) {
		anisotropy[i] = (double**)malloc(XSIZE * sizeof(double *));
		for(j = 0; j<RADIUS;j++) {
			anisotropy[i][j] = (double*)malloc(RADIUS * sizeof(double));
		}
	}

	//Initializing 3D matrix anzimuth
	double*** azimuth;
	azimuth = (double***)malloc(YSIZE * sizeof(double**));
	for(i = 0;i<YSIZE;i++) {
		azimuth[i] = (double**)malloc(XSIZE * sizeof(double *));
		for(j = 0; j<RADIUS;j++) {
			azimuth[i][j] = (double*)malloc(RADIUS * sizeof(double));
		}
	}
	
	


	//Actual computation
	int xrad,yrad,x,y,k,index1,cor_bi_MinInd;
	double tempCompute,tempSum,cor_bi_ColMin;
	//for (y=0;y<YSIZE;y++) {

		//if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS + 1))) continue;
	y = 0;
	for(x = 0;x<XSIZE+1;x++) {
		//printf("%d\n",y);
		//printf("Loop1\n");
		if(x==XSIZE) {
			y++;
			if(y==YSIZE){
				x = XSIZE;
				continue;
			}
			x=0;
			continue;
			
		}
		
		if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS + 1))) continue;
		printf("Loop2\n");
		if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS + 1))) continue;	
		printf("Loop3\n");

		for(j = 0;j<RADIUS;j+=RADSTEP) {
			for(i=0;i<ANGLESIZE;i++) {
				xrad = (int)round(cos(angle[i]) * (j+1) + x);	//<------------IT WORKS; VERIFIED
				yrad = (int)round(sin(angle[i]) * (j+1) + y);	//<------------IT WORKS; VERIFIED
	//			printf("%d) x %d	y %d\n",i+1,xrad,yrad);			
	//			printf("\t %d %d \n",(int)round(cos(angle[i]) * (j+1) + x),(int)round(sin(angle[i]) * (j+1) + y));

				cmatrix[i][j] = (double)data[yrad-1][xrad-1]; 	//<------------IT WORKS; VERIFIED
	//			printf("%d) xrad %d	yrad %d	data %f	data-1 %f\n",i+1,xrad,yrad,(double)data[yrad][xrad],(double)data[yrad-1][xrad-1]);	
	//			printf("%d) %f\n",i+1,cmatrix[i][j]);
				tempSum = 0;
				tempCompute = 0;

				for(index1 = 0;index1<=j;index1++) {					
					tempCompute = cmatrix[i][index1] - (double)data[y-1][x-1];
				//	printf("%d,%d	CM %f	DA%f\n",x,y,cmatrix[i][index1],(double)data[y-1][x-1]);
					tempCompute  = tempCompute * tempCompute;
				//	tempCompute = tempCompute / (2*j);
					tempSum = tempSum + tempCompute;
					//printf("%d,i %d,j %d) CM %f	DA %f	TS %f	",index1,i,j,cmatrix[i][index1],(double)data[y][x],tempSum);
				}
				
				cor[i][j] = tempSum/(2*(j+1));	//<------------IT WORKS; VERIFIED
				//printf("cor %f\n",tempSum/(2*(j+1)));
				printf("%d) %f\n",i+1,cor[i][j]);
				//printf("%f \n",tempSum);
			}
			return 0;
			
			cor_bi_ColMin = DBL_MAX;
			cor_bi_MinInd = 0;
			for (k=0;k<(ANGLESIZE)/2;k++) {
				cor_bi[k][j] = (cor[k][j] + cor[k+36][j])/2;
				if(cor_bi[k][j] < cor_bi_ColMin) {
					cor_bi_ColMin = cor_bi[k][j];
					cor_bi_MinInd = k;
					//printf("%f,%d\n",cor_bi_ColMin,cor_bi_MinInd);	
				}
				
			}
			int tmp;
			/*for(k=0;k<72;k++) {
				for(tmp=0;tmp<100;tmp++) {
						printf("%f ",cor[k][tmp]);
				}
				printf("\n");

			}*/
			
			
		}


		
	}
	
	//}

	//printf("%f",DBL_MAX);

	fclose(datTxt);

	//Freeing matrix cor
	for(i=0;i<ANGLESIZE;i++){
		free(cor[i]);
	}
	free(cor);

	//Freeing matrix cor_bi
	for(i=0;i<ANGLESIZE/2;i++){
		free(cor_bi[i]);
	}
	free(cor_bi);
	
	//Freeing matrix cmatrix
	for(i=0;i<ANGLESIZE;i++){
		free(cmatrix[i]);
	}
	free(cmatrix);

	//Freeing 3D matrix anisotropy
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(anisotropy[i][j]);
		}
		free(anisotropy[i]);
	}
	free(anisotropy);

	//Freeing 3D matrix azimuth
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(azimuth[i][j]);
		}
		free(azimuth[i]);
	}
	free(azimuth);
	
	return 0;
}
