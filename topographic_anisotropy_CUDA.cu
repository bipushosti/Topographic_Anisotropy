

#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define XSIZE 	1201
#define YSIZE	801



#define RADIUS		100
#define	RADSTEP		1
#define ANGLESIZE	72	


#define PI 3.14

//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* cmatrix,float* cor,float* cor_bi,float* angle,float* anisotropy,float* azimuth);

//--------------------------------------------------------------------------------------------------------------------------//



__global__ void getMatrix(int* data,float* cmatrix,float* cor,float* cor_bi,float* angle,float* anisotropy,float* azimuth)
{

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
//	int index = col + row * RADIUS/RADSTEP;

	//int x = col + 100;
	//int y = row + 100;
	if((y<701)&&(y>100) && (x<1101)&&(x> 100)) {

		int xrad,yrad,i,j,k,index1,cor_bi_MinInd;
		xrad = 0;
		yrad = 0;

		//float cmatrix[ANGLESIZE][RADIUS/RADSTEP];
		//float cor[ANGLESIZE][RADIUS/RADSTEP];
		//float cor_bi[ANGLESIZE/2][RADIUS/RADSTEP];


		float tempSum,tempCompute,cor_bi_ColMin,cor_bi_Ortho;

		for(j = 0;j<RADIUS;j+=RADSTEP) {
			for(i=0;i<ANGLESIZE;i++) {
				xrad = (int)round(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)round(sinf(angle[i]) * (j+1) + y);	

				cmatrix[i * RADIUS/RADSTEP + j] = (float)data[(yrad-1 ) * XSIZE + xrad-1]; 	

				tempSum = 0;
				tempCompute = 0;

				for(index1 = 0;index1<=j;index1++) {					
					tempCompute = cmatrix[i * RADIUS/RADSTEP + index1] - (float)data[(y-1) * XSIZE + x-1];
					tempCompute  = tempCompute * tempCompute ;
					tempSum = (tempSum + tempCompute);
				}
		
				cor[i * RADIUS/RADSTEP + j] = tempSum/(2*(j+1));	
			}
	
			cor_bi_ColMin = FLT_MAX;
			cor_bi_MinInd = 0;
			cor_bi_Ortho = 0;
			for (k=0;k<(ANGLESIZE)/2;k++) {

				cor_bi[k * RADIUS/RADSTEP + j] = (cor[k * RADIUS/RADSTEP + j] + cor[(k+36) * RADIUS/RADSTEP + j])/2 ;

				if(cor_bi[k *RADIUS/RADSTEP + j] < cor_bi_ColMin) {					
					cor_bi_ColMin = cor_bi[k *RADIUS/RADSTEP + j];
					cor_bi_MinInd = k;
				}
			}

			if(cor_bi_MinInd <18) {								
				cor_bi_Ortho = cor_bi[(cor_bi_MinInd + 18)* RADIUS/RADSTEP + j];
			}
			else {
				cor_bi_Ortho = cor_bi[(cor_bi_MinInd - 18)*RADIUS/RADSTEP + j];
			}		

			//Fail safe to ensure there is no nan or inf			
			if(cor_bi_ColMin == 0) {
					if((cor_bi_Ortho < 1) && (cor_bi_Ortho > 0)) {
						cor_bi_ColMin = cor_bi_Ortho;
					}
					else {
						cor_bi_ColMin = 1;
					}
			}

			if(cor_bi_Ortho == 0) {
				cor_bi_Ortho = 1;
			}
		}
		
		anisotropy[y * YSIZE * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = cor_bi_Ortho/cor_bi_ColMin;
		azimuth[y * YSIZE * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = angle[cor_bi_MinInd] * 180/PI ;	
		
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//




int main()
{
	FILE *datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	int data[YSIZE][XSIZE];

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}
	//1200 ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char line[1200 * 5 +2+1200];
	memset(line, '\0', sizeof(line));
	char *startPtr,*endPtr;
	
	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}

	outputAnisotropy00 = fopen("outputDataAni00.txt","w");
	outputAnisotropy09 = fopen("outputDataAni09.txt","w");
	outputAnisotropy49 = fopen("outputDataAni49.txt","w");
	outputAnisotropy99 = fopen("outputDataAni99.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi00.txt","w");
	outputAzimuth09 = fopen("outputDataAzi09.txt","w");
	outputAzimuth49 = fopen("outputDataAzi49.txt","w");
	outputAzimuth99 = fopen("outputDataAzi99.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}

	int i,j,Value;
	j = 0;
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	while(fgets(line,1200 *5 + 2 + 1200,datTxt)!=NULL) {	
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d ",Value);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d\n",Value);
			}
		}
		
		j++;
	}	
	//return 0;

	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
		//printf("%d	::	%f\n",i,angle[i]);
	}
	

	//Initializing 2D cmatrix
	float** cmatrix;
	cmatrix = (float**)malloc(ANGLESIZE * sizeof(float*));
	for(i=0;i<ANGLESIZE;i++) {
		cmatrix[i] = (float*)malloc(RADIUS/RADSTEP *sizeof(float));
	}

	//Initializing cor
	float** cor;
	cor = (float**)malloc(ANGLESIZE * sizeof(float*));
	for(i=0;i<ANGLESIZE;i++) {
		cor[i] = (float*)malloc(RADIUS/RADSTEP *sizeof(float));
	}

	//Initializing cor_bi
	float** cor_bi;
	cor_bi = (float**)malloc(ANGLESIZE/2 * sizeof(float*));
	for(i=0;i<ANGLESIZE/2;i++) {
		cor_bi[i] = (float*)malloc(RADIUS/RADSTEP *sizeof(float));
	}

	//Initializing 3D matrix anisotropy
	float*** anisotropy;
	anisotropy = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		anisotropy[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			anisotropy[i][j] = (float*)malloc(RADIUS/RADSTEP * sizeof(float));
		}
	}

	//Initializing 3D matrix anzimuth
	float*** azimuth;
	azimuth = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		azimuth[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			azimuth[i][j] = (float*)malloc(RADIUS/RADSTEP * sizeof(float));
		}
	}
	
	


	//Actual computation
	int xrad,yrad,x,y,k,index1,cor_bi_MinInd;
	float tempCompute,tempSum,cor_bi_ColMin,cor_bi_Ortho;

//--------------------------------------CUDA-----------------------------------------------------//



	int* ptrD_data,*ptrH_data;
	float* ptrD_angle,*ptrD_cmatrix,*ptrD_cor,*ptrD_corbi,*ptrD_azimuth,*ptrD_anisotropy;
	float *ptrH_anisotropy,*ptrH_azimuth,*ptrH_angle,*ptrH_cor,*ptrH_corbi,*ptrH_cmatrix;
	ptrH_angle = &angle[0];
	ptrH_data = &data[0][0];
	ptrH_cmatrix = &cmatrix[0][0];
	ptrH_cor = &cor[0][0];
	ptrH_corbi = &cor_bi[0][0];
	ptrH_azimuth = &azimuth[0][0][0];
	ptrH_anisotropy = &anisotropy[0][0][0];

	hipMalloc((void**)&ptrD_data,XSIZE * YSIZE * sizeof(int));
	hipMalloc((void**)&ptrD_cmatrix,ANGLESIZE * RADIUS/RADSTEP * sizeof(float));
	hipMalloc((void**)&ptrD_cor,ANGLESIZE * RADIUS/RADSTEP * sizeof(float));
	hipMalloc((void**)&ptrD_corbi,ANGLESIZE/2 * RADIUS/RADSTEP * sizeof(float));

	hipMalloc((void**)&ptrD_angle,ANGLESIZE * sizeof(float));
	hipMalloc((void**)&ptrD_azimuth,YSIZE * XSIZE * RADIUS/RADSTEP * sizeof(float));
	hipMalloc((void**)&ptrD_anisotropy,YSIZE * XSIZE * RADIUS/RADSTEP * sizeof(float));

	hipMemcpy(ptrD_data,ptrH_data,XSIZE * YSIZE * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(ptrD_cmatrix,ptrH_cmatrix,ANGLESIZE * RADIUS/RADSTEP * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(ptrD_cor,ptrH_cor,ANGLESIZE *RADIUS/RADSTEP * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(ptrD_corbi,ptrH_corbi,ANGLESIZE/2 * RADIUS/RADSTEP * sizeof(float),hipMemcpyHostToDevice);

	hipMemcpy(ptrD_azimuth,ptrH_azimuth,YSIZE * XSIZE * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(ptrD_anisotropy,ptrH_anisotropy,XSIZE*YSIZE* sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(ptrD_angle,angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice);
	printf("Hello\n");


	const dim3 gridSize(38,YSIZE,1);
	const dim3 blockSize(32,1,1);
	
	getMatrix<<<gridSize,blockSize>>>(ptrD_data,ptrD_cmatrix,ptrD_cor,ptrD_corbi,ptrD_angle,ptrD_anisotropy,ptrD_azimuth);

	printf("Hello2\n");


	hipMemcpy(ptrH_azimuth,ptrD_azimuth,YSIZE * XSIZE * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(ptrH_anisotropy,ptrD_anisotropy,YSIZE * XSIZE * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);


	printf("Hello3\n");
	
	printf("Hello4\n");
//	cudaFree(ptrH_anisotropy);
//	cudaFree(ptrH_azimuth);

	hipFree(ptrD_data);
	hipFree(ptrD_angle);
	hipFree(ptrD_azimuth);
	hipFree(ptrD_anisotropy);
	
	hipFree(ptrD_cmatrix);
	hipFree(ptrD_cor);
	hipFree(ptrD_corbi);

//------------------------------------------------------------------------------------------------//
//			Writing to files


/*
	for(j = 0;j<YSIZE;j++) {
			for(i=0;i<XSIZE;i++) {

				if (i == (XSIZE - RADIUS - 1)) {
					fprintf(outputAnisotropy00,"%f\n",anisotropy[i][j][0]);
				}
				else {
					fprintf(outputAnisotropy00,"%f\t",anisotropy[i][j][0]);
				}
			}

	}
*/
	printf("Hello5\n");


	/*if(j == 0) {
		if (x == (XSIZE - RADIUS - 1)) {
			fprintf(outputAnisotropy00,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth00,"%f",azimuth[y][x][j]);
			fprintf(outputAnisotropy00,"\n");
			fprintf(outputAzimuth00,"\n");
		}
		else {
			fprintf(outputAnisotropy00,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth00,"%f",azimuth[y][x][j]);
			fprintf(outputAnisotropy00,"\t");
			fprintf(outputAzimuth00,"\t");
		}
	}

	else if(j == 9) {
		if (x == (XSIZE - RADIUS - 1)) {
			fprintf(outputAnisotropy09,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth09,"%f",azimuth[y][x][j]);
			fprintf(outputAnisotropy09,"\n");
			fprintf(outputAzimuth09,"\n");
		}
		else {
			fprintf(outputAnisotropy09,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth09,"%f",azimuth[y][x][j]);
			fprintf(outputAnisotropy09,"\t");
			fprintf(outputAzimuth09,"\t");
		}
	}
	else if(j == 49) {
	
		if (x == (XSIZE - RADIUS - 1)) {
			fprintf(outputAnisotropy49,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth49,"%f",azimuth[y][x][j]);
			fprintf(outputAnisotropy49,"\n");
			fprintf(outputAzimuth49,"\n");
		}
		else {
			fprintf(outputAnisotropy49,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth49,"%f",azimuth[y][x][j]);	
			fprintf(outputAnisotropy49,"\t");
			fprintf(outputAzimuth49,"\t");
		}
	}
	else if(j == 99) {
	
		if (x == (XSIZE - RADIUS - 1)) {
			fprintf(outputAnisotropy99,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth99,"%f",azimuth[y][x][j]);
			fprintf(outputAnisotropy99,"\n");
			fprintf(outputAzimuth99,"\n");
		}
		else {
			fprintf(outputAnisotropy99,"%f",anisotropy[y][x][j]);
			fprintf(outputAzimuth99,"%f",azimuth[y][x][j]);
			fprintf(outputAnisotropy99,"\t");
			fprintf(outputAzimuth99,"\t");
		}
	}

	//}

	//printf("%f",DBL_MAX);
*/
	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);
	


	printf("%f\n",anisotropy[0][0][0]);

	//Freeing matrix cor
	for(i=0;i<ANGLESIZE;i++){
		free(cor[i]);
	}
	free(cor);

	//Freeing matrix cor_bi
	for(i=0;i<ANGLESIZE/2;i++){
		free(cor_bi[i]);
	}
	free(cor_bi);
	
	//Freeing matrix cmatrix
	for(i=0;i<ANGLESIZE;i++){
		free(cmatrix[i]);
	}
	free(cmatrix);

//------------------Works only when this is commented out!!---------------//
//------------------Strange as the matrices have to be freed!-------------//

	//Freeing 3D matrix anisotropy
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(anisotropy[i][j]);
		}
		free(anisotropy[i]);
	}
	free(anisotropy);

	//Freeing 3D matrix azimuth
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(azimuth[i][j]);
		}
		free(azimuth[i]);
	}
	free(azimuth);


//	free(ptrH_anisotropy);
//	free(ptrH_azimuth);
	return 0;
}
