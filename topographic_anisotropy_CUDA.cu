

#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>


#define GPU_MEMSIZE_GB		2

#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280
#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 

//#define XSIZE 		1201
//#define YSIZE			801



#define RADIUS			100
#define	RADSTEP			1
#define ANGLESIZE		36	


#define PI 3.14

//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,size_t XSIZE,size_t YSIZE);

//--------------------------------------------------------------------------------------------------------------------------//

//Current Usage:
//Global Memory: 773 MB
//Memory per Thread: 1.28 KiloBytes
//	Thread Memory Usage 	=Total Threads * Memory Per Thread 
//				= 1001 * 601 * 1.28KB
//				= 770.05 MB

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,size_t XSIZE,size_t YSIZE)
{
	
	//Actual computation
	int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
	int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;
	float variance[100];
	float orientation[100];
	float ortho[100];
	float value,sum_value,avg_value;
	float sum_valueOrtho,avg_valueOrtho;
	sum_value = 0;
	avg_value = 0;
	sum_valueOrtho = 0;
	avg_valueOrtho = 0;

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
	else
	{

		
		for(i=0;i<100;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
		}
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			//Initializing to 0 so that the sum is zero everytime it starts
			sum_value = 0;
			sum_valueOrtho = 0;
			for(j = 0;j<RADIUS;j+=RADSTEP) {
	
				//Computation for angle of interest
				xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

				value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
				value = value * value;
				
				//One eighty angle computation
				xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
				yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
				valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
				valueOneEighty = valueOneEighty * valueOneEighty;

				sum_value = sum_value + value + valueOneEighty;
				avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

				//Computation for values on angle orthogonal to angle of interest
				xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
				yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
				valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
				valueOrtho1 = valueOrtho1 * valueOrtho1;

				//One eighty ortho angle computation
				xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
				yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

				valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
				valueOrtho2 = valueOrtho2 * valueOrtho2;

				sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
				avg_valueOrtho = sum_valueOrtho/(2*j+1);

				//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
				if(avg_value == 0) {
						if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
							avg_value = avg_valueOrtho;
						}
						else {
							avg_value = 1;
						}
				}

				if(avg_valueOrtho == 0) {
					avg_valueOrtho = 1;
				}
				
				//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
				if(avg_value < variance[j]) {
						variance[j] = avg_value;
						orientation[j] = angle[i];
						ortho[j] = avg_valueOrtho;		
				}	
			}
		}
		for(j=0;j<RADIUS;j+=RADSTEP){	
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = ortho[j]/variance[j];
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[j] * 180/PI;
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//




int main()
{
	
	

	//Setting the output buffer to 500MB
	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);

	//File declarations and opening them
	FILE *datTxt1,*datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}
	
	
	datTxt1 = fopen("dat.txt","r");
	if(datTxt1 == NULL) {
		printf("Cannot open dat.txt file\n");
		exit(1);
	}
	outputAnisotropy00 = fopen("outputDataAni00.txt","w");
	outputAnisotropy09 = fopen("outputDataAni09.txt","w");
	outputAnisotropy49 = fopen("outputDataAni49.txt","w");
	outputAnisotropy99 = fopen("outputDataAni99.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi00.txt","w");
	outputAzimuth09 = fopen("outputDataAzi09.txt","w");
	outputAzimuth49 = fopen("outputDataAzi49.txt","w");
	outputAzimuth99 = fopen("outputDataAzi99.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}


//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//

	int XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line)if(*max_line++ == ' ')++XSIZE;
	XSIZE+=1;
	
	


	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	fclose(datTxt1);

	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		printf("Cannot open dat.txt file\n");
		exit(1);
	}
//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//

	printf("(XSIZE,YSIZE):(%d,%d)\n",XSIZE,YSIZE);
	//printf("Maximum size possible = %f\nTotal size of current data(XSIZE * YSIZE) = %zd\n",MAX_XSIZE_POSSIBLE,XSIZE * YSIZE);
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//

	int data[YSIZE][XSIZE];

	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[(XSIZE-1) * 5 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	int Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 5 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	printf("Working1\n");
	while(fgets(line,(XSIZE-1) *5 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		printf("Working2\n");
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d ",Value);
				printf("%d\n",Value);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d\n",Value);
			}
		}
		
		j++;
	}	
	
	
//------------------------------------Matrix Declarations--------------------------------------------------------------------------------------------------------------//
	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
		//printf("%d	::	%f\n",i,angle[i]);
	}

	float* anisotropy;
	anisotropy = (float*)malloc(YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
	float *azimuth;
	azimuth = (float*)malloc(YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));

	//anisotropy[0][0][99] = 834;
	
	
//--------------------------------------CUDA-------------------------------------------------------------------------------------------------------------------------//



	int *data_ptr;
	float *anisotropy_ptr,*azimuth_ptr,*angle_ptr;
/*
	float *anisotropy_ptrH;
	anisotropy_ptrH = &anisotropy[0][0][0];

	float *azimuth_ptrH;
	azimuth_ptrH = &azimuth[0][0][0];
*/

	hipMalloc((void**)&data_ptr,XSIZE * YSIZE * sizeof(int));
	hipMemcpy(data_ptr,data,XSIZE * YSIZE * sizeof(int),hipMemcpyHostToDevice);

	hipMalloc((void**)&angle_ptr,ANGLESIZE * sizeof(float));
	hipMemcpy(angle_ptr,angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice);
	
	hipMalloc((void**)&anisotropy_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
	hipMalloc((void**)&azimuth_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));


	printf("Hello1\n");

	dim3 gridSize(3,YSIZE,1);
	dim3 blockSize(512,1,1);

	printf("Hello2\n");

	getMatrix<<<gridSize,blockSize>>>(data_ptr,angle_ptr,anisotropy_ptr,azimuth_ptr,XSIZE,YSIZE);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
  	{
		printf("CUDA Error: %s\n", hipGetErrorString(error));

    	// we can't recover from the error -- exit the program
    	return 0;
  	}

	printf("Hello3\n");
	
	hipMemcpy(anisotropy,anisotropy_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(azimuth,azimuth_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);
	

	printf("Hello4\n");

	hipFree(data_ptr);
	hipFree(angle_ptr);
	hipFree(azimuth_ptr);
	hipFree(anisotropy_ptr);
	printf("Hello5\n");

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
//			Writing to files


	for(j=0;j<YSIZE ;j++) {
		for(i=0;i<XSIZE ;i++) {
			if((j>(YSIZE - RADIUS - 1))||(j<(RADIUS))) continue;
			if((i>(XSIZE - RADIUS - 1))||(i<(RADIUS))) continue;

			if (i == (XSIZE  - RADIUS - 1)) {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\n");
				fprintf(outputAzimuth00,"\n");

				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAnisotropy09,"\n");
				fprintf(outputAzimuth09,"\n");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);
				fprintf(outputAnisotropy49,"\n");
				fprintf(outputAzimuth49,"\n");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAnisotropy99,"\n");
				fprintf(outputAzimuth99,"\n");
			}
			else {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\t");
				fprintf(outputAzimuth00,"\t");
	
				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAnisotropy09,"\t");
				fprintf(outputAzimuth09,"\t");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);	
				fprintf(outputAnisotropy49,"\t");
				fprintf(outputAzimuth49,"\t");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAnisotropy99,"\t");
				fprintf(outputAzimuth99,"\t");	
			}					
		}
	}	


	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);
	

	
	//free(max_line);
	free(anisotropy);
	free(azimuth);

	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte );
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;
	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	return 0;
}
