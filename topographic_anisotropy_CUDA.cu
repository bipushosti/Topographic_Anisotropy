

#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>



#define XSIZE 	1201
#define YSIZE	801



#define RADIUS		100
#define	RADSTEP		1
#define ANGLESIZE	36	


#define PI 3.14

//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth);

//--------------------------------------------------------------------------------------------------------------------------//



__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth)
{
	
	//Actual computation
	int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
	int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;
	float variance[100];
	float orientation[100];
	float ortho[100];
	float value,sum_value,avg_value;
	float sum_valueOrtho,avg_valueOrtho;
	sum_value = 0;
	avg_value = 0;
	sum_valueOrtho = 0;
	avg_valueOrtho = 0;

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int index = x + y * RADIUS/RADSTEP;
	

//	if((y<701)&&(y>99) && (x<1101)&&(x>99))
	if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
//	if((y>(801 - 100 - 1))||(y<(100))||(x>(1201 - 100 - 1))||(x<(100))) return;
	else
	{

		printf("X=%d,Y=%d\n",x,y);
		for(i=0;i<100;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
		}
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			//Initializing to 0 so that the sum is zero everytime it starts
			sum_value = 0;
			sum_valueOrtho = 0;
			for(j = 0;j<RADIUS;j+=RADSTEP) {
	
				//Computation for angle of interest
				xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

				value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
				value = value * value;
				
				//One eighty angle computation
				xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
				yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
				valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
				valueOneEighty = valueOneEighty * valueOneEighty;

				sum_value = sum_value + value + valueOneEighty;
				avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

				//Computation for values on angle orthogonal to angle of interest
				xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
				yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
				valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
				valueOrtho1 = valueOrtho1 * valueOrtho1;

				//One eighty ortho angle computation
				xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
				yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

				valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
				valueOrtho2 = valueOrtho2 * valueOrtho2;

				sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
				avg_valueOrtho = sum_valueOrtho/(2*j+1);

				//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
				if(avg_value == 0) {
						if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
							avg_value = avg_valueOrtho;
						}
						else {
							avg_value = 1;
						}
				}

				if(avg_valueOrtho == 0) {
					avg_valueOrtho = 1;
				}
				
				//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
				if(avg_value < variance[j]) {
					//	printf("2(%d)	%f	%f\n",j,variance[j],avg_value);
						variance[j] = avg_value;
						orientation[j] = angle[i];
						ortho[j] = avg_valueOrtho;		
				}	
			}
		}
		//__syncthreads();
		for(j=0;j<RADIUS;j+=RADSTEP){
			//atomicExch(&anisotropy[y * YSIZE * XSIZE + x * RADIUS/RADSTEP + j], ortho[j]/variance[j]);
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = ortho[j]/variance[j];
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[j] * 180/PI;
			//atomicExch(&azimuth[y * YSIZE * XSIZE + x * RADIUS/RADSTEP + j] , orientation[j] * 180/PI);
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//




int main()
{
	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);
	printf("Limit is %u\n",(unsigned)limit);


	FILE *datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	int data[YSIZE][XSIZE];

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}
	//1200 ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char line[1200 * 5 +2+1200];
	memset(line, '\0', sizeof(line));
	char *startPtr,*endPtr;
	
	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}

	outputAnisotropy00 = fopen("outputDataAni00.txt","w");
	outputAnisotropy09 = fopen("outputDataAni09.txt","w");
	outputAnisotropy49 = fopen("outputDataAni49.txt","w");
	outputAnisotropy99 = fopen("outputDataAni99.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi00.txt","w");
	outputAzimuth09 = fopen("outputDataAzi09.txt","w");
	outputAzimuth49 = fopen("outputDataAzi49.txt","w");
	outputAzimuth99 = fopen("outputDataAzi99.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}

	int i,j,Value;
	j = 0;
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	while(fgets(line,1200 *5 + 2 + 1200,datTxt)!=NULL) {	
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d ",Value);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d\n",Value);
			}
		}
		
		j++;
	}	
	//return 0;

//------------------------------------Matrix Declarations------------------------------------------//
	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
		//printf("%d	::	%f\n",i,angle[i]);
	}
/*	
	//Initializing 3D matrix anisotropy
	float*** anisotropy;
	anisotropy = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		anisotropy[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			anisotropy[i][j] = (float*)malloc(RADIUS/RADSTEP * sizeof(float));
		}
	}


	//Initializing 3D matrix anzimuth
	float*** azimuth;
	azimuth = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		azimuth[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			azimuth[i][j] = (float*)malloc(RADIUS/RADSTEP * sizeof(float));
		}
	}

*/
	float* anisotropy;
	anisotropy = (float*)malloc(YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
	float *azimuth;
	azimuth = (float*)malloc(YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));

	//anisotropy[0][0][99] = 834;
	
	
//--------------------------------------CUDA-----------------------------------------------------//



	int *data_ptr;
	float *anisotropy_ptr,*azimuth_ptr,*angle_ptr;
/*
	float *anisotropy_ptrH;
	anisotropy_ptrH = &anisotropy[0][0][0];

	float *azimuth_ptrH;
	azimuth_ptrH = &azimuth[0][0][0];
*/

	hipMalloc((void**)&data_ptr,XSIZE * YSIZE * sizeof(int));
	hipMemcpy(data_ptr,data,XSIZE * YSIZE * sizeof(int),hipMemcpyHostToDevice);

	hipMalloc((void**)&angle_ptr,ANGLESIZE * sizeof(float));
	hipMemcpy(angle_ptr,angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice);
	
	hipMalloc((void**)&anisotropy_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
	hipMalloc((void**)&azimuth_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float));


	printf("Hello1\n");

	dim3 gridSize(3,YSIZE,1);
	dim3 blockSize(512,1,1);

	printf("Hello2\n");

	getMatrix<<<gridSize,blockSize>>>(data_ptr,angle_ptr,anisotropy_ptr,azimuth_ptr);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
  	{
		printf("CUDA Error: %s\n", hipGetErrorString(error));

    	// we can't recover from the error -- exit the program
    	return 0;
  	}

	printf("Hello3\n");
	
	hipMemcpy(anisotropy,anisotropy_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(azimuth,azimuth_ptr,YSIZE  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost);
	

	printf("Hello4\n");

	hipFree(data_ptr);
	hipFree(angle_ptr);
	hipFree(azimuth_ptr);
	hipFree(anisotropy_ptr);
	printf("Hello5\n");

//------------------------------------------------------------------------------------------------//
//			Writing to files


	for(j=0;j<YSIZE ;j++) {
		for(i=0;i<XSIZE ;i++) {
			if((j>(YSIZE - RADIUS - 1))||(j<(RADIUS))) continue;
			if((i>(XSIZE - RADIUS - 1))||(i<(RADIUS))) continue;

			if (i == (XSIZE  - RADIUS - 1)) {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\n");
				fprintf(outputAzimuth00,"\n");

				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAnisotropy09,"\n");
				fprintf(outputAzimuth09,"\n");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);
				fprintf(outputAnisotropy49,"\n");
				fprintf(outputAzimuth49,"\n");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAnisotropy99,"\n");
				fprintf(outputAzimuth99,"\n");
			}
			else {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\t");
				fprintf(outputAzimuth00,"\t");
	
				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 9]);
				fprintf(outputAnisotropy09,"\t");
				fprintf(outputAzimuth09,"\t");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 49]);	
				fprintf(outputAnisotropy49,"\t");
				fprintf(outputAzimuth49,"\t");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 99]);
				fprintf(outputAnisotropy99,"\t");
				fprintf(outputAzimuth99,"\t");	
			}					
		}
	}	


	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);
	

/*
	//Freeing 3D matrix anisotropy
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(anisotropy[i][j]);
		}
		free(anisotropy[i]);
	}
	free(anisotropy);

	//Freeing 3D matrix azimuth
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(azimuth[i][j]);
		}
		free(azimuth[i]);
	}
	free(azimuth);
*/

	free(anisotropy);
	free(azimuth);


	return 0;
}
