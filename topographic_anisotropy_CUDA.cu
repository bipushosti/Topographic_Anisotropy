


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define DATASIZE 	792134
//The number of chars in x,y or z vars
#define VARSIZE		13

#define RADIUS		1800
#define	RADWINDOW	4000
#define	RADSTEP		900
#define ANGLESIZE	72	

#define LEFT	9.892e5
#define RIGHT	1.191e6
#define UP		3.011e6
#define DOWN	2.85e6

#define PI 3.141592653589793

int main()
{
	FILE *datTxt;
	//Has to be allocated since too big
	double *x,*y,*z;
	x = (double *)malloc(sizeof(double) * DATASIZE);
	y = (double *)malloc(sizeof(double) * DATASIZE);
	z = (double *)malloc(sizeof(double) * DATASIZE);

	//Max needed is VARSIZE + 6 as there are 3 spaces between 
	//vars.But 15 in case the format changes.
	char line[VARSIZE *3 + 15];
	memset(line, '\0', sizeof(line));
	char *startPtr,*endPtr;
	
	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}
	int i;
	char tempX[VARSIZE],tempY[VARSIZE],tempZ[VARSIZE];
	memset(tempX, '\0', sizeof(tempX));
	memset(tempY, '\0', sizeof(tempY));
	memset(tempZ, '\0', sizeof(tempZ));

	while(fgets(line,VARSIZE*3+10,datTxt)!=NULL) {
		//printf("%d\n",sizeof(line)/sizeof(char));
		startPtr = line;
		for(i=0;i<2;i++) {
			endPtr = strchr(startPtr,' ');
			printf("Hello\n");
			if(endPtr != NULL) {
				printf("Hello1\n");
				if(i==0) {
					strncpy(tempX,startPtr,endPtr-startPtr); 
					printf("Hello2\n");
				}
				else 
					strncpy(tempY,startPtr,endPtr-startPtr); 
					printf("Hello3\n");
			}
			
			endPtr = endPtr + 3;
			startPtr = endPtr;
			
		}
		break;
	//	endPtr = strchr(line,'\0');
	//	if(endPtr != NULL) {
	//		strncpy(tempZ,startPtr,endPtr-startPtr); 
	//	}

		
	}	

	//fclose(datTxt);
/*
	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
//		printf("%f",angle[i]);
	}
	float cor[ANGLESIZE][RADIUS/RADSTEP];
	float cor_bi[ANGLESIZE/2][ANGLESIZE];
	//Setting cor to be all zeros
	memset(cor,0,sizeof(cor[0][0]) * ANGLESIZE * RADIUS/RADSTEP);
	memset(cor_bi,0,sizeof(cor_bi[0][0])* ANGLESIZE/2 * ANGLESIZE);
	printf("%f",cor[0][0]);
*/
	fclose(datTxt);
	free(x);
	free(y);
	free(z);
	return 0;
}
