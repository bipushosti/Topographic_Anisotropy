


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define XSIZE 	1200
#define YSIZE	800


#define RADIUS		100
//#define	RADWINDOW	4000
#define	RADSTEP		1
#define ANGLESIZE	72	

#define LEFT	9.892e5
#define RIGHT	1.191e6
#define UP		3.011e6
#define DOWN	2.85e6

#define PI 3.141592653589793

int main()
{
	FILE *datTxt;
	int data[YSIZE][XSIZE];

	//1200 ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char line[1200 * 5 +2+1200];
	memset(line, '\0', sizeof(line));
	char *startPtr,*endPtr;
	
	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}

	int i,j,Value;
	j = 0;
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	while(fgets(line,1200 *5 + 2 + 1200,datTxt)!=NULL) {
	
		startPtr = line;
		//Skipping the first space
		startPtr+=1;
		//printf("%s",startPtr);

		
		for(i=0;i<XSIZE;i++) {
			endPtr = strchr(startPtr,' ');
			if(endPtr != NULL) {	
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j][i] = Value;
			}	
			endPtr = endPtr + 1;
			startPtr = endPtr;	
		}
		j++;
	}	

	

	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
	}



	//double cor[ANGLESIZE][RADIUS/RADSTEP];
	double** cor;
	cor = (double**)malloc(ANGLESIZE * sizeof(double*));
	for(i=0;i<ANGLESIZE;i++) {
		cor[i] = (double*)malloc(RADIUS/RADSTEP *sizeof(double));
	}

	
	double** cor_bi;
	cor_bi = (double**)malloc(ANGLESIZE/2 * sizeof(double*));
	for(i=0;i<ANGLESIZE/2;i++) {
		cor_bi[i] = (double*)malloc(RADIUS/RADSTEP *sizeof(double));
	}


	double*** anisotropy;
	anisotropy = (double***)malloc(YSIZE * sizeof(double**));
	for(i = 0;i<YSIZE;i++) {
		anisotropy[i] = (double**)malloc(XSIZE * sizeof(double *));
		for(j = 0; j<RADIUS;j++) {
			anisotropy[i][j] = (double*)malloc(RADIUS * sizeof(double));
		}
	}
	//double anisotropy[YSIZE][XSIZE][RADIUS];
	//float azimuth[YSIZE][XSIZE][RADIUS];


	fclose(datTxt);

	//Freeing matrix cor
	for(i=0;i<ANGLESIZE;i++){
		free(cor[i]);
	}
	free(cor);

	//Freeing matrix cor_bi
	for(i=0;i<ANGLESIZE/2;i++){
		free(cor_bi[i]);
	}
	free(cor_bi);
	
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(anisotropy[i][j]);
		}
		free(anisotropy[i]);
	}
	free(anisotropy);
	



	return 0;
}
