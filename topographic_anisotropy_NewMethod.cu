
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <limits.h>

#define XSIZE 	1201
#define YSIZE	801



#define RADIUS		100
#define	RADSTEP		1
#define ANGLESIZE	72	


#define PI 3.141592653589793

int main()
{
	FILE *datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	int data[YSIZE][XSIZE];

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}
	//1200 ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char line[1200 * 5 +2+1200];
	memset(line, '\0', sizeof(line));
	char *startPtr,*endPtr;
	
	datTxt = fopen("dat.txt","r");
	if(datTxt == NULL) {
		perror("Cannot open dat.txt file");
		return (-1);
	}

	outputAnisotropy00 = fopen("outputDataAni00.txt","w");
	outputAnisotropy09 = fopen("outputDataAni09.txt","w");
	outputAnisotropy49 = fopen("outputDataAni49.txt","w");
	outputAnisotropy99 = fopen("outputDataAni99.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi00.txt","w");
	outputAzimuth09 = fopen("outputDataAzi09.txt","w");
	outputAzimuth49 = fopen("outputDataAzi49.txt","w");
	outputAzimuth99 = fopen("outputDataAzi99.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}

	int i,j,Value;
	j = 0;
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	while(fgets(line,1200 *5 + 2 + 1200,datTxt)!=NULL) {	
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d ",Value);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j][i] = Value;
				fprintf(inpCheck,"%d\n",Value);
			}
		}
		
		j++;
	}	

	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
	}
	
	//Initializing 3D matrix anisotropy
	float*** anisotropy;
	anisotropy = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		anisotropy[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			anisotropy[i][j] = (float*)malloc(RADIUS * sizeof(float));
		}
	}

	//Initializing 3D matrix anzimuth
	float*** azimuth;
	azimuth = (float***)malloc(YSIZE * sizeof(float**));
	for(i = 0;i<YSIZE;i++) {
		azimuth[i] = (float**)malloc(XSIZE * sizeof(float *));
		for(j = 0; j<XSIZE;j++) {
			azimuth[i][j] = (float*)malloc(RADIUS * sizeof(float));
		}
	}

	//Actual computation
	int xrad,yrad,x,y,xradOrtho,yradOrtho;
	float variance[100];
	float orientation[100];
	float ortho[100];
	float value,sum_value,avg_value;
	float valueOrtho,sum_valueOrtho,avg_valueOrtho;
	sum_value = 0;
	avg_value = 0;
	sum_valueOrtho = 0;
	avg_valueOrtho = 0;

	

	//y = 0;
	for(y=0;y<YSIZE;y++) {
		for(x = 0;x<XSIZE;x++) {
		/*for(x = 0;x<XSIZE+1;x++) {
			if(x==XSIZE) {
				y++;	
				if(y==YSIZE){
					x = XSIZE;
					continue;
				}
				x=0;
				continue;
			
			}
			*/
			if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS + 1))) continue;
			if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS + 1))) continue;	

			for(i=0;i<100;i++){
				variance[i] = FLT_MAX;
			}

			//Flipped
			for(i=0;i<ANGLESIZE;i++) {
				sum_value = 0;
				for(j = 0;j<RADIUS;j+=RADSTEP) {
		
					xrad = (int)round(cos(angle[i]) * (j+1) + x);	
					yrad = (int)round(sin(angle[i]) * (j+1) + y);	


					value = data[y][x] - data[yrad][xrad];
					value = (value * value)*0.5;
					
					sum_value = sum_value + value;
					avg_value = sum_value/(j+1);

					//printf("1(%d,%d)	%f	%f\n",(j+1),(i+1),variance[j],avg_value);
					if(avg_value < variance[j]) {
					//	printf("2(%d)	%f	%f\n",j,variance[j],avg_value);
						variance[j] = avg_value;
						orientation[j] = angle[i];

					//	Ortho computation
						xradOrtho = (int)round(cos(angle[i]) * (j+1) + x);	
						yradOrtho = (int)round(sin(angle[i]) * (j+1) + y);	

						valueOrtho = data[y][x] - data[yradOrtho][xradOrtho];
						valueOrtho = (value * value)*0.5;
					
						sum_valueOrtho = sum_valueOrtho + valueOrtho;
						avg_valueOrtho = sum_valueOrtho/(j+1);

						ortho[j] = avg_valueOrtho;		
					}	
				}
			}
			
			for(j=0;j<RADIUS;j+=RADSTEP){
				anisotropy[y][x][j] = variance[j];
				azimuth[y][x][j] = orientation[j] * 180/PI ;
				
				//printf("%f	%f\n",variance[j],anisotropy[y][x][j]);	
			}

			
//			Writing to files
			

			if (x == (XSIZE - RADIUS - 1)) {
				fprintf(outputAnisotropy00,"%f",anisotropy[y][x][0]);
				fprintf(outputAzimuth00,"%f",azimuth[y][x][0]);
				fprintf(outputAnisotropy00,"\n");
				fprintf(outputAzimuth00,"\n");

				fprintf(outputAnisotropy09,"%f",anisotropy[y][x][9]);
				fprintf(outputAzimuth09,"%f",azimuth[y][x][9]);
				fprintf(outputAnisotropy09,"\n");
				fprintf(outputAzimuth09,"\n");

				fprintf(outputAnisotropy49,"%f",anisotropy[y][x][49]);
				fprintf(outputAzimuth49,"%f",azimuth[y][x][49]);
				fprintf(outputAnisotropy49,"\n");
				fprintf(outputAzimuth49,"\n");

				fprintf(outputAnisotropy99,"%f",anisotropy[y][x][99]);
				fprintf(outputAzimuth99,"%f",azimuth[y][x][99]);
				fprintf(outputAnisotropy99,"\n");
				fprintf(outputAzimuth99,"\n");
			}
			else {
				fprintf(outputAnisotropy00,"%f",anisotropy[y][x][0]);
				fprintf(outputAzimuth00,"%f",azimuth[y][x][0]);
				fprintf(outputAnisotropy00,"\t");
				fprintf(outputAzimuth00,"\t");
				
				fprintf(outputAnisotropy09,"%f",anisotropy[y][x][9]);
				fprintf(outputAzimuth09,"%f",azimuth[y][x][9]);
				fprintf(outputAnisotropy09,"\t");
				fprintf(outputAzimuth09,"\t");

				fprintf(outputAnisotropy49,"%f",anisotropy[y][x][49]);
				fprintf(outputAzimuth49,"%f",azimuth[y][x][49]);	
				fprintf(outputAnisotropy49,"\t");
				fprintf(outputAzimuth49,"\t");

				fprintf(outputAnisotropy99,"%f",anisotropy[y][x][99]);
				fprintf(outputAzimuth99,"%f",azimuth[y][x][99]);
				fprintf(outputAnisotropy99,"\t");
				fprintf(outputAzimuth99,"\t");
				
			}					
		}			
			
	}


		
	
	


	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);

	//Freeing 3D matrix anisotropy
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(anisotropy[i][j]);
		}
		free(anisotropy[i]);
	}
	free(anisotropy);

	//Freeing 3D matrix azimuth
	for(i = 0;i<YSIZE;i++) {
		for(j=0;j<XSIZE;j++) {
			free(azimuth[i][j]);
		}
		free(azimuth[i]);
	}
	free(azimuth);
	
	return 0;
}
