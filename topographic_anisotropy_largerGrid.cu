#include "hip/hip_runtime.h"




//Input file: space delimited

#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "topographic_anisotropy_largerGrid.h"

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Size of the GPU memory
#define GPU_MEMSIZE_GB		2

//For case in which XSIZE = 1201 and YSIZE = 801
#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280

//MAX_XSIZE_POSSIBLE is the maximum size of x or max number of columns if there is only one row
#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 


//#define XSIZE 		1201
//#define YSIZE			801


//Always have even number of radius;and divisible by 10
#define RADIUS			100
#define	RADSTEP			1
#define ANGLESIZE		36	//Size of angle array	

#define PI 			3.14159


#define THREADS_PER_BLOCK	512

//#define FILENAME	"Annie_coastDEM.txt"
//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,int XSIZE,int YSIZE);
int Get_GPU_devices();
static void HandleError(hipError_t err,const char *file, int line);
//--------------------------------------------------------------------------------------------------------------------------//

//Current Usage:
//Global Memory: 773 MB


__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,int XSIZE,int YSIZE)
{
//	SGR I don't see where XSIZE or YSIZE are defined...
/*	
	//Actual computation
	int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
	int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

//	Hardwired to be at 100 Radius now. This needs to change!
	float variance[100];
	float orientation[100];
	float ortho[100];


	float value,sum_value,avg_value;
	float sum_valueOrtho,avg_valueOrtho;

//	Initializing declared variables
	sum_value = 0;
	avg_value = 0;
	sum_valueOrtho = 0;
	avg_valueOrtho = 0;
*/


	//for(int i = 0;i <YSIZE*XSIZE;i++){
	//	printf("%d\n",data[i]);
	//}

//	Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//----------------------------------------------------------------------------------------------------------------------------//	
	if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
	else
	{
		//printf("%d,%d\n",XSIZE,YSIZE);
		//Actual computation
		int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
		int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

	
		float variance[RADIUS];
		float orientation[RADIUS];
		float ortho[RADIUS];


		float value,sum_value,avg_value;
		float sum_valueOrtho,avg_valueOrtho;

	//	Initializing declared variables
		sum_value = 0;
		avg_value = 0;
		sum_valueOrtho = 0;
		avg_valueOrtho = 0;

	//	Iniitalize variance, ortho, and orientation arrays with max float value SGR changed i<100 to i<RADIUS
		for(i=0;i<RADIUS;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
		}
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			//Initializing to 0 so that the sum is zero everytime it starts
			sum_value = 0;
			sum_valueOrtho = 0;

			for(j = 0;j<RADIUS;j+=RADSTEP) {
	
				//Computation for angle of interest
				xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

				value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
				value = value * value;
				
				//One eighty angle computation
				xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
				yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
				valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
				valueOneEighty = valueOneEighty * valueOneEighty;

				sum_value = sum_value + value + valueOneEighty;
				avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

				//Computation for values on angle orthogonal to angle of interest
				xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
				yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
				valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
				valueOrtho1 = valueOrtho1 * valueOrtho1;

				//One eighty ortho angle computation
				xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
				yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

				valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
				valueOrtho2 = valueOrtho2 * valueOrtho2;

				sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
				avg_valueOrtho = sum_valueOrtho/(2*j+1);

				//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
				if(avg_value == 0) {
					if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
						avg_value = avg_valueOrtho;
					}
					else{
						avg_value = 1;
					}
				}

				if(avg_valueOrtho == 0) {
					avg_valueOrtho = 1;
				}
				
				//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
				if(avg_value < variance[j]) {
					variance[j] = avg_value;
					orientation[j] = angle[i];
					ortho[j] = avg_valueOrtho;		
				}	
			}
		}

		for(j=0;j<RADIUS;j+=RADSTEP){	
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = ortho[j]/variance[j];
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[j] * 180/PI;
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//


//--------------------------------------Handle Error()-----------------------------------------------------------//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


//--------------------------------------Get_GPU_devices()-----------------------------------------------------------//

int Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));

	if(!prop.deviceOverlap){
		printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		printf("%d Devices Found\n",DeviceCount);
	}else{
		printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	for(int i = 0;i<DeviceCount;i++){
		hipDeviceProp_t properties;
		HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", properties.name);
		printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
		printf("\n");

	}

	return DeviceCount;
}
//#################################################################################################################################//
//--------------------------------------Main()-----------------------------------------------------------//
int main(int argc,char* argv[])
//int main()
{
	if(argc == 1){
		printf("Not enough arguments\n");
		return 0;
	}
		
//-----------Getting the number of devices and their sizes------------------------------------------------//
        int DeviceCount;
        DeviceCount = Get_GPU_devices();

	GPU_struct	GPU_values[DeviceCount];

	#undef RADIUS
	#define RADIUS atoi(argv[2])
	//Setting the output buffer to 500MB
	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);

	//File declarations and opening them
	FILE *datTxt1,*datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open inpcheck.txt file");
		return (-1);
	}
	

	datTxt1 = fopen(argv[1],"r");	
	//datTxt1 = fopen("Annie_coastDEM.txt","r");
	if(datTxt1 == NULL) {
		printf("Cannot open file: %s  \nCheck if file exists.\n",argv[1]);
		exit(1);
	}
	outputAnisotropy00 = fopen("outputDataAni_First.txt","w");
	outputAnisotropy09 = fopen("outputDataAni_Rad_div_10.txt","w");
	outputAnisotropy49 = fopen("outputDataAni_Rad_div_2.txt","w");
	outputAnisotropy99 = fopen("outputDataAni_Last.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi_First.txt","w");
	outputAzimuth09 = fopen("outputDataAzi_Rad_div_10.txt","w");
	outputAzimuth49 = fopen("outputDataAzi_Rad_div_2.txt","w");
	outputAzimuth99 = fopen("outputDataAzi_Last.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}


//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//

	int XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	long int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line)if(*max_line++ == ' ')++XSIZE;
	XSIZE+=1;
	
	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	fclose(datTxt1);
	printf("(XSIZE,YSIZE)::(%d,%d)\n",XSIZE,YSIZE);

	datTxt = fopen(argv[1],"r");
//	datTxt = fopen("Annie_coastDEM.txt","r");
	if(datTxt == NULL) {
		//printf("Cannot open file: %s\nCheck if file exists\n",argv[1]);
		exit(1);
	}
//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//

	printf("(XSIZE,YSIZE):(%d,%d)\n",XSIZE,YSIZE);
	//printf("Maximum size possible = %f\nTotal size of current data(XSIZE * YSIZE) = %zd\n",MAX_XSIZE_POSSIBLE,XSIZE * YSIZE);
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");




//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
	int data[YSIZE * XSIZE];

//	int* data;

//	data = (int*)malloc(YSIZE * XSIZE * sizeof(int));

	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[XSIZE * 10 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	int Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 7 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	printf("Working1\n");
	while(fgets(line,XSIZE *10 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		//printf("Working2\n");
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d ",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d\n",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);
			}
		}
		
		j++;
	}	
	printf("Done data[%zd][%zd] = %d\n",j-1,i-1,data[500 * XSIZE + 500]);	
	printf("Working File IO\n");

//----------Getting number of rows for each GPU-------------------------------------------------------------------------------------------------------------//
	//Variable that holds YSIZE initially. This changes as number of rows 
	//for each GPU is calculated
	int tmpSize = 0;
	//Variable needed to compute the total rows each GPU will have
	int count = 0;
	//size holds the total size of the data for each of the parts
	size_t size = 0;
	//size_orig holds the original size without the RADIUS added to it multiplied by RADIUS/RADSTEP
	size_t size_orig = 0;
	//offset holds either 2*RADIUS or RADIUS depending on the part of data
	int offset = 0;
	//sum of the total positions of the rows for each GPU in each iteration
	int pos = 0;
	//Actual position inside the data matrix
	size_t data_position = 0;
	//Store the total size of the angle array in bytes
	size_t angle_bytes = ANGLESIZE * sizeof(float);

	tmpSize = YSIZE;
	count = DeviceCount;

	printf("Tmpsize = %zd,Device Count = %d\n",tmpSize,DeviceCount);
	//Iterating through all the available devices
	for(i = 0;i<DeviceCount;i++){

		printf("\n########################Device %d #############################\n",i);

		if(tmpSize % count != 0){
			GPU_values[i].NumRows = (tmpSize/count) + 1;
			GPU_values[i].NumCols = XSIZE;
		}else{
			GPU_values[i].NumRows = tmpSize/count;
			GPU_values[i].NumCols = XSIZE;
		}	
		//Values change here as the num of rows for each gpu is 
		//calculated after each iteration
		tmpSize = tmpSize - GPU_values[i].NumRows;
		count--;

		printf("Row Value is: %d\n",GPU_values[i].NumRows);

		size_orig = GPU_values[i].NumRows * XSIZE * RADIUS/RADSTEP *sizeof(float);

		if((i == 0) ||(i == (DeviceCount -1))){
			size = (GPU_values[i].NumRows + RADIUS ) * XSIZE * sizeof(float);	

			//Setting the offset into the data matrix
			if(i == 0){
				offset = 0;
			}else if(i == (DeviceCount - 1)){
				offset = RADIUS * -1;
			}
		//Sections in between
		}else{
			size = (GPU_values[i].NumRows + 2*RADIUS) * XSIZE * sizeof(float);
			offset = RADIUS * -1;
		}
		
		printf("Size is: GPU_values[%zd].NumRows + RADIUS = (%d + %d )*%d *%d =  %ld\n",i,GPU_values[i].NumRows,RADIUS,XSIZE,sizeof(float),size);

		//-----------------Matrix Allocations----------------------------//
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipStreamCreate(&GPU_values[i].stream));

		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_anisotropy,size_orig));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_azimuth,size_orig));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_data,size));	
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_angle,angle_bytes));
		printf("Cuda Malloc to GPU possible \n");


		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_anisotropy,size_orig));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_azimuth,size_orig));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_data,size));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_angle,angle_bytes));	

		printf("Cuda Malloc to CPU possible \n");

		
//---------------Initialization of data arrays for each GPU---------// 
		//Initially pos = 0 and offset = 0 so "if" statement is unnecessary

		for(j=0;j<ANGLESIZE;j++) {
			GPU_values[i].h_angle[j] = j * 5 * PI/180;		
		}


		size_t total_data;
		total_data = size/sizeof(float);

		printf("Total data size is %ld\n",total_data);

		printf("The size of the array is %d\n",sizeof(GPU_values[i].h_data));

		//if(i == 0){		
			data_position = (pos + offset) * XSIZE;
		//}else{
		//	data_position = (pos + offset + 1) * XSIZE;
		//}
		
		printf("pos = %d,data_position after sub Index = %zd\n",pos,data_position);
		//Initializing the data arrays in each of the gpu with portions of the main data
		for(j=0;j<(total_data);j++){
			GPU_values[i].h_data[j] = data[data_position+j];
			//if(i==1){
				//if(j!=0 && j % 501 == 0) printf("\n");
				//printf("%d ",GPU_values[i].h_data[j]);			
			//}
		}
		
		for(j=0;j<(total_data);j++){
				if((j!=0) && (j % 501 == 0)) printf("\n");
				printf("%d ",GPU_values[i].h_data[j]);			
			
		}
		printf("\n\n\n\n\n\n");
		



		printf("Data array assigned \n");
		offset = RADIUS * -1;
		//if(i == 0) {
		//	pos+=GPU_values[i].NumRows;
		//}else {
			pos+=GPU_values[i].NumRows;
		//}
	}
	printf("Working3\n");
	
//----------------Sending data to GPU;Launching Kernel and getting data back-----------------------//
	for(i=0;i<DeviceCount;i++){
		
		HANDLE_ERROR(hipSetDevice(i));
		//HANDLE_ERROR(hipStreamSynchronize(GPU_values[i].stream));

		if((i == 0) ||(i == (DeviceCount -1))){
			size = (GPU_values[i].NumRows + RADIUS) * XSIZE * sizeof(float);			
		//Sections in between
		}else{
			size = (GPU_values[i].NumRows + 2*RADIUS) * XSIZE * sizeof(float);
		}
//-----------------Sending data to GPU----------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_data,GPU_values[i].h_data,size,hipMemcpyHostToDevice,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_angle,GPU_values[i].h_angle,angle_bytes,hipMemcpyHostToDevice,GPU_values[i].stream));


//----------------Launching the Kernel---------------------//
		dim3 gridSize((XSIZE + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK ,YSIZE,1);
		dim3 blockSize(THREADS_PER_BLOCK,1,1);

		getMatrix<<<gridSize,blockSize,0,GPU_values[i].stream>>>(GPU_values[i].d_data,GPU_values[i].d_angle,GPU_values[i].d_anisotropy,GPU_values[i].d_azimuth,XSIZE,YSIZE);
//---------------Getting data back------------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_anisotropy,GPU_values[i].d_anisotropy,size_orig,hipMemcpyDeviceToHost,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_azimuth,GPU_values[i].d_azimuth,size_orig,hipMemcpyDeviceToHost,GPU_values[i].stream));
	}
//------------------------------------Freeing data-----------------------------------------------------------------------//
	for(i=0;i<DeviceCount;i++){
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_anisotropy));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_azimuth));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_data));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_angle));


		HANDLE_ERROR(hipFree(GPU_values[i].d_anisotropy));
		HANDLE_ERROR(hipFree(GPU_values[i].d_azimuth));
		HANDLE_ERROR(hipFree(GPU_values[i].d_data));
		
		HANDLE_ERROR(hipStreamDestroy(GPU_values[i].stream));
		hipDeviceReset();
	}

	fclose(datTxt);
	fclose(datTxt1);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);
	
//	free(data);
	
	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte );
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;
	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	return 0;
}
		
