#include "hip/hip_runtime.h"




//Input file: space delimited

#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Size of the GPU memory
#define GPU_MEMSIZE_GB		2

//For case in which XSIZE = 1201 and YSIZE = 801
#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280

//MAX_XSIZE_POSSIBLE is the maximum size of x or max number of columns if there is only one row
#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 


//#define XSIZE 		1201
//#define YSIZE			801


//Always have even number of radius;and divisible by 10
#define RADIUS			100
#define	RADSTEP			1
#define ANGLESIZE		36	//Size of angle array	

#define PI 3.14


#define THREADS_PER_BLOCK	512

//#define FILENAME	"Annie_coastDEM.txt"
//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,int XSIZE,int YSIZE);
int Get_GPU_devices();
static void HandleError(hipError_t err,const char *file, int line);
//--------------------------------------------------------------------------------------------------------------------------//

//Current Usage:
//Global Memory: 773 MB


__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,int XSIZE,int YSIZE)
{
//	SGR I don't see where XSIZE or YSIZE are defined...
/*	
	//Actual computation
	int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
	int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

//	Hardwired to be at 100 Radius now. This needs to change!
	float variance[100];
	float orientation[100];
	float ortho[100];


	float value,sum_value,avg_value;
	float sum_valueOrtho,avg_valueOrtho;

//	Initializing declared variables
	sum_value = 0;
	avg_value = 0;
	sum_valueOrtho = 0;
	avg_valueOrtho = 0;
*/


	//for(int i = 0;i <YSIZE*XSIZE;i++){
	//	printf("%d\n",data[i]);
	//}

//	Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//----------------------------------------------------------------------------------------------------------------------------//	
	if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
	else
	{
		//printf("%d,%d\n",XSIZE,YSIZE);
		//Actual computation
		int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
		int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

	
		float variance[RADIUS];
		float orientation[RADIUS];
		float ortho[RADIUS];


		float value,sum_value,avg_value;
		float sum_valueOrtho,avg_valueOrtho;

	//	Initializing declared variables
		sum_value = 0;
		avg_value = 0;
		sum_valueOrtho = 0;
		avg_valueOrtho = 0;

	//	Iniitalize variance, ortho, and orientation arrays with max float value SGR changed i<100 to i<RADIUS
		for(i=0;i<RADIUS;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
		}
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			//Initializing to 0 so that the sum is zero everytime it starts
			sum_value = 0;
			sum_valueOrtho = 0;

			for(j = 0;j<RADIUS;j+=RADSTEP) {
	
				//Computation for angle of interest
				xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

				value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
				value = value * value;
				
				//One eighty angle computation
				xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
				yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
				valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
				valueOneEighty = valueOneEighty * valueOneEighty;

				sum_value = sum_value + value + valueOneEighty;
				avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

				//Computation for values on angle orthogonal to angle of interest
				xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
				yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
				valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
				valueOrtho1 = valueOrtho1 * valueOrtho1;

				//One eighty ortho angle computation
				xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
				yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

				valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
				valueOrtho2 = valueOrtho2 * valueOrtho2;

				sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
				avg_valueOrtho = sum_valueOrtho/(2*j+1);

				//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
				if(avg_value == 0) {
					if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
						avg_value = avg_valueOrtho;
					}
					else{
						avg_value = 1;
					}
				}

				if(avg_valueOrtho == 0) {
					avg_valueOrtho = 1;
				}
				
				//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
				if(avg_value < variance[j]) {
					variance[j] = avg_value;
					orientation[j] = angle[i];
					ortho[j] = avg_valueOrtho;		
				}	
			}
		}

		for(j=0;j<RADIUS;j+=RADSTEP){	
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = ortho[j]/variance[j];
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[j] * 180/PI;
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//


//--------------------------------------Handle Error()-----------------------------------------------------------//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


//--------------------------------------Get_GPU_devices()-----------------------------------------------------------//

int Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));

	if(!prop.deviceOverlap){
		printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		printf("%d Devices Found\n",DeviceCount);
	}else{
		printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	for(int i = 0;i<DeviceCount;i++){
		hipDeviceProp_t properties;
		HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", properties.name);
		printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
		printf("\n");

	}

	return DeviceCount;
}
//--------------------------------------Main()-----------------------------------------------------------//
int main(int argc,char* argv[])
//int main()
{
	if(argc == 1){
		printf("Not enough arguments\n");
		return 0;
	}
		
	#undef RADIUS
	#define RADIUS atoi(argv[2])
	//Setting the output buffer to 500MB
	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);

	//File declarations and opening them
	FILE *datTxt1,*datTxt,*outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth49,*outputAzimuth99; 
	

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open inpcheck.txt file");
		return (-1);
	}
	

	datTxt1 = fopen(argv[1],"r");	
	//datTxt1 = fopen("Annie_coastDEM.txt","r");
	if(datTxt1 == NULL) {
		printf("Cannot open file: %s  \nCheck if file exists.\n",argv[1]);
		exit(1);
	}
	outputAnisotropy00 = fopen("outputDataAni_First.txt","w");
	outputAnisotropy09 = fopen("outputDataAni_Rad_div_10.txt","w");
	outputAnisotropy49 = fopen("outputDataAni_Rad_div_2.txt","w");
	outputAnisotropy99 = fopen("outputDataAni_Last.txt","w");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen("outputDataAzi_First.txt","w");
	outputAzimuth09 = fopen("outputDataAzi_Rad_div_10.txt","w");
	outputAzimuth49 = fopen("outputDataAzi_Rad_div_2.txt","w");
	outputAzimuth99 = fopen("outputDataAzi_Last.txt","w");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}


//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//

	int XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line)if(*max_line++ == ' ')++XSIZE;
	XSIZE+=1;
	
	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	fclose(datTxt1);
	printf("(XSIZE,YSIZE)::(%d,%d)\n",XSIZE,YSIZE);

	datTxt = fopen(argv[1],"r");
//	datTxt = fopen("Annie_coastDEM.txt","r");
	if(datTxt == NULL) {
		//printf("Cannot open file: %s\nCheck if file exists\n",argv[1]);
		exit(1);
	}
//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//

	printf("(XSIZE,YSIZE):(%d,%d)\n",XSIZE,YSIZE);
	//printf("Maximum size possible = %f\nTotal size of current data(XSIZE * YSIZE) = %zd\n",MAX_XSIZE_POSSIBLE,XSIZE * YSIZE);
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");




//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
	int data[YSIZE * XSIZE];

	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[XSIZE * 10 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	int Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 5 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	printf("Working1\n");
	while(fgets(line,XSIZE *10 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		//printf("Working2\n");
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d ",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				data[j * XSIZE + i] = Value;
				fprintf(inpCheck,"%d\n",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);
			}
		}
		
		j++;
	}	
	
//-----------Getting the number of devices and their sizes---------------------------------------------------------------------------------------------------//
	int DeviceCount;	
	DeviceCount = Get_GPU_devices();

//----------Getting number of rows for each GPU-------------------------------------------------------------------------------------------------------------//

	int tmpSize = 0;
	int count = 0;
	//GPU_sizes holds the number of rows each of the GPU devices will hold 
	int GPU_sizes[DeviceCount];
	tmpSize = YSIZE;
	count = DeviceCount;

	int* data_ptr[DeviceCount];
	float *anisotropy_ptr[DeviceCount],*azimuth_ptr[DeviceCount],*angle_ptr;
	
	for(int i = 0;i<DeviceCount;i++){
		if(tmpSize % count != 0){
			GPU_sizes[i] = (tmpSize/count) + 1;
		}else{
			GPU_sizes[i] = tmpSize/count;
		}	
		tmpSize = tmpSize - GPU_sizes[i];
		count--;
		printf("%d\n",GPU_sizes[i]);
	}

//------------------------------------Matrix Declarations-----------------------------------------------------------------------------------------------------//
	float angle[ANGLESIZE];
	for(int i=0;i<ANGLESIZE;i++) {
		angle[i] = i * 5 * PI/180;
		//printf("%d	::	%f\n",i,angle[i]);
	}

	float* anisotropy[DeviceCount],*azimuth[DeviceCount];

	int dev = 0;
	int pos = 0;

	//Allocating memory according to the section of the data
	for(dev = 0; dev<DeviceCount; dev++){
		if((dev == 0) ||(dev == (DeviceCount -1))){
			anisotropy[dev] = (float*)malloc((GPU_sizes[dev]+RADIUS)  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
			azimuth[dev] = (float*)malloc((GPU_sizes[dev]+RADIUS)  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
		}else{
			anisotropy[dev] = (float*)malloc((GPU_sizes[dev]+2*RADIUS)  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
			azimuth[dev] = (float*)malloc((GPU_sizes[dev]+2*RADIUS)  * XSIZE  * RADIUS/RADSTEP * sizeof(float));
		}
	}

//----------Dividing and allocating memory for the data in GPU and sending data-----------------------------------------------------------------------------------------//

	//size holds the total size of the data for each of the parts
	size_t size = 0;
	//offset holds either 2*RADIUS or RADIUS depending on the part of data
	int offset = 0;

	for(dev = 0,pos=0;dev<DeviceCount; pos+=GPU_sizes[dev],dev++){

		hipSetDevice(dev);
		//If the first section or last
		if((dev == 0) ||(dev == (DeviceCount -1))){
			size = (GPU_sizes[dev] + RADIUS) * XSIZE * sizeof(float);
			
		//Sections in between
		}else{
			size = (GPU_sizes[dev] + 2*RADIUS) * XSIZE * sizeof(float);
		}

		//Setting the offset into the data matrix
		if(dev == 0){
			offset = 0;
		}else{
			offset = RADIUS * -1;
		}



		//hipMalloc((void**)&data_ptr[dev],GPU_sizes[dev] * XSIZE * sizeof(float));
		HANDLE_ERROR(hipMalloc((void**)&data_ptr[dev],size));
		HANDLE_ERROR(hipMalloc((void**)&angle_ptr,ANGLESIZE * sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&anisotropy_ptr[dev],size * RADIUS/RADSTEP));
		HANDLE_ERROR(hipMalloc((void**)&azimuth_ptr[dev],size  * RADIUS/RADSTEP));

//CHECKED!!----Getting the correct values from the data array	
		//for (int j = 0;j<size/sizeof(float);j++){
		//	printf("%d\n",*(data + XSIZE* (pos+offset)+j));
		//}
		//printf("----------------------------------------------------------------\n");
		//printf("Size: %zd\n",size/sizeof(float));		

		//return 0;

//ERROR HERE!!
		HANDLE_ERROR(hipMemcpy(data_ptr[dev],(pos+offset) * XSIZE + data,size,hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(angle_ptr,angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice));
	}

/*
//----------Sending data to the GPU-----------------------------------------------------------------------------------------------------------------------------//
	//int pos = 0;
	for(dev = 0,pos=0;dev<DeviceCount; pos+=GPU_sizes[dev],dev++){
		hipSetDevice(dev);
		if((dev == 0) ||(dev == (DeviceCount -1))){
			size = (GPU_sizes[dev] + RADIUS) * XSIZE * sizeof(float);
		}else{
			size = (GPU_sizes[dev] + 2*RADIUS) * XSIZE * sizeof(float);
		}
		hipMemcpy(data_ptr[dev],pos * XSIZE + data,size,hipMemcpyHostToDevice);
		hipMemcpy(angle_ptr,angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice);

		//hipMemcpy(anisotropy_ptr[dev],pos * XSIZE * RADIUS/RADSTEP + data,GPU_sizes[dev] * XSIZE * RADIUS/RADSTEP * sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(azimuth_ptr[dev],pos * XSIZE * RADIUS/RADSTEP + data,GPU_sizes[dev] * XSIZE * RADIUS/RADSTEP * sizeof(float),hipMemcpyHostToDevice);
	}
*/

//--------------------------------------CUDA--------------------------------------------------------------------------------------------------------------------//
	//dim3 gridSize((XSIZE + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK ,YSIZE,1);
	dim3 blockSize(THREADS_PER_BLOCK,1,1);

	printf("Hello2\n");
	int rows_y = 0;



	//for(int i = 0;i <251*XSIZE;i++){
	//	printf("%d\n",data[i]);
	//}



	for(dev = 0; dev < DeviceCount; dev++){

		hipSetDevice(dev);
		if((dev == 0) ||(dev == (DeviceCount -1))){
			rows_y = GPU_sizes[dev] + RADIUS;
		//Sections in between
		}else{
			rows_y = GPU_sizes[dev] + 2 * RADIUS;
		}
		dim3 gridSize((XSIZE + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK ,rows_y,1);
		getMatrix<<<gridSize,blockSize>>>(data_ptr[dev],angle_ptr,anisotropy_ptr[dev],azimuth_ptr[dev],XSIZE,rows_y);
	}

	printf("Hello3\n");
	//hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
  	{
		printf("Error\n");
		printf("CUDA Error: %s\n", hipGetErrorString(error));

    	// we can't recover from the error -- exit the program
    	return 0;
  	}

	
	rows_y = 0;	
	for(dev = 0; dev < DeviceCount;dev++){
		hipSetDevice(dev);
		if((dev == 0) ||(dev == (DeviceCount -1))){
			rows_y = GPU_sizes[dev] + RADIUS;
		//Sections in between
		}else{
			rows_y = GPU_sizes[dev] + 2 * RADIUS;
		}

//For error checking only 
/*
		HANDLE_ERROR(hipMalloc((void**)&anisotropy_ptr[dev],size * RADIUS/RADSTEP));
		HANDLE_ERROR(hipMalloc((void**)&azimuth_ptr[dev],size  * RADIUS/RADSTEP));
*/
		HANDLE_ERROR(hipMemcpy(anisotropy[dev],anisotropy_ptr[dev],rows_y  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(azimuth[dev],azimuth_ptr[dev],rows_y  * XSIZE  * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost));
	}

	printf("Hello4\n");

	for(dev = 0; dev<DeviceCount; dev++){

		hipFree(data_ptr[i]);	
		
		hipFree(azimuth_ptr[i]);
		hipFree(anisotropy_ptr[i]);
	}
	hipFree(angle_ptr);

	printf("%f\n",*(anisotropy[0]+1));
	printf("Hello5\n");

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//
//			Writing to files
/*

	for(j=0;j<YSIZE ;j++) {
		for(i=0;i<XSIZE ;i++) {
			if((j>(YSIZE - RADIUS - 1))||(j<(RADIUS))) continue;
			if((i>(XSIZE - RADIUS - 1))||(i<(RADIUS))) continue;

			if (i == (XSIZE  - RADIUS - 1)) {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\n");
				fprintf(outputAzimuth00,"\n");

				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP +RADIUS/10 -1]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAnisotropy09,"\n");
				fprintf(outputAzimuth09,"\n");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAnisotropy49,"\n");
				fprintf(outputAzimuth49,"\n");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS -1]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAnisotropy99,"\n");
				fprintf(outputAzimuth99,"\n");
			}
			else {
				fprintf(outputAnisotropy00,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAzimuth00,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
				fprintf(outputAnisotropy00,"\t");
				fprintf(outputAzimuth00,"\t");
	
				fprintf(outputAnisotropy09,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAzimuth09,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/10 -1]);
				fprintf(outputAnisotropy09,"\t");
				fprintf(outputAzimuth09,"\t");

				fprintf(outputAnisotropy49,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
				fprintf(outputAzimuth49,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);	
				fprintf(outputAnisotropy49,"\t");
				fprintf(outputAzimuth49,"\t");

				fprintf(outputAnisotropy99,"%f",anisotropy[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAzimuth99,"%f",azimuth[j * XSIZE  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
				fprintf(outputAnisotropy99,"\t");
				fprintf(outputAzimuth99,"\t");	
			}					
		}
	}	

*/
	fclose(datTxt);
	fclose(inpCheck);
	fclose(outputAnisotropy00);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth09);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);
	

	
	//free(max_line);
	for(dev = 0; dev<DeviceCount;dev++){
		free(anisotropy[dev]);
		free(azimuth[dev]);
	}
	
	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte );
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;
	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	return 0;
}
