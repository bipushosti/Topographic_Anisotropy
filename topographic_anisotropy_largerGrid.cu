#include "hip/hip_runtime.h"





//Input file: space delimited

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "topographic_anisotropy_largerGrid.h"

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Size of the GPU memory
#define GPU_MEMSIZE_GB		2

//For case in which XSIZE = 1201 and YSIZE = 801
#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280

//MAX_XSIZE_POSSIBLE is the maximum size of x or max number of columns if there is only one row
#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 


//#define XSIZE 		1201
//#define YSIZE			801


//Always have even number of radius;and divisible by 10
#define RADIUS			100
#define	RADSTEP			1
#define ANGLESIZE		36	//Size of angle array	

#define PI 			3.14159


#define THREADS_PER_BLOCK	512

//#define FILENAME	"Annie_coastDEM.txt"
//---------------------------Function declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,int XSIZE,int YSIZE);
int Get_GPU_devices();
static void HandleError(hipError_t err,const char *file, int line);
inline hipError_t checkCuda(hipError_t result);
//--------------------------------------------------------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,int XSIZE,int YSIZE)
{

//	Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//----------------------------------------------------------------------------------------------------------------------------//	
	if((y>(YSIZE - RADIUS - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
	else
	{
		//printf("%d,%d\n",XSIZE,YSIZE);
		//Actual computation
		int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
		int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

	
		float variance[RADIUS];
		float orientation[RADIUS];
		float ortho[RADIUS];


		float value,sum_value,avg_value;
		float sum_valueOrtho,avg_valueOrtho;

	//	Initializing declared variables
		sum_value = 0;
		avg_value = 0;
		sum_valueOrtho = 0;
		avg_valueOrtho = 0;

	//	Iniitalize variance, ortho, and orientation arrays with max float value SGR changed i<100 to i<RADIUS
		for(i=0;i<RADIUS;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
		}
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			//Initializing to 0 so that the sum is zero everytime it starts
			sum_value = 0;
			sum_valueOrtho = 0;

			for(j = 0;j<RADIUS;j+=RADSTEP) {
	
				//Computation for angle of interest
				xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

				value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
				value = value * value;
				
				//One eighty angle computation
				xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
				yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
				valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
				valueOneEighty = valueOneEighty * valueOneEighty;

				sum_value = sum_value + value + valueOneEighty;
				avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

				//Computation for values on angle orthogonal to angle of interest
				xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
				yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
				valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
				valueOrtho1 = valueOrtho1 * valueOrtho1;

				//One eighty ortho angle computation
				xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
				yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

				valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
				valueOrtho2 = valueOrtho2 * valueOrtho2;

				sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
				avg_valueOrtho = sum_valueOrtho/(2*j+1);

				//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
				if(avg_value == 0) {
					if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
						avg_value = avg_valueOrtho;
					}
					else{
						avg_value = 1;
					}
				}

				if(avg_valueOrtho == 0) {
					avg_valueOrtho = 1;
				}
				
				//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
				if(avg_value < variance[j]) {
					variance[j] = avg_value;
					orientation[j] = angle[i];
					ortho[j] = avg_valueOrtho;		
				}	
			}
		}

		for(j=0;j<RADIUS;j+=RADSTEP){	
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = ortho[j]/variance[j];
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[j] * 180/PI;
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//



//--------------------------------------Handle Error()-----------------------------------------------------------//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


//--------------------------------------Get_GPU_devices()-----------------------------------------------------------//

int Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));

	if(!prop.deviceOverlap){
		printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		printf("%d Devices Found\n",DeviceCount);
	}else{
		printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	for(int i = 0;i<DeviceCount;i++){
		hipDeviceProp_t properties;
		HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", properties.name);
		printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
		printf("\n");

	}

	return DeviceCount;
}



//#################################################################################################################################//
//#################################################################################################################################//
//#################################################################################################################################//

//--------------------------------------Main()-----------------------------------------------------------//
int main(int argc,char* argv[])
//int main()
{


	if(argc == 1){
		printf("Not enough arguments\n");
		return 0;
	}
		


	#undef RADIUS
	#define RADIUS atoi(argv[2])
	//Setting the output buffer to 500MB
	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);

	//File declarations and opening them
	FILE *datTxt1,*datTxt;

	

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open inpcheck.txt file");
		return (-1);
	}
	

	datTxt1 = fopen(argv[1],"r");	
	if(datTxt1 == NULL) {
		printf("Cannot open file: %s  \nCheck if file exists.\n",argv[1]);
		exit(1);
	}



//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//

	int XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	long int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line)if(*max_line++ == ' ')++XSIZE;
	XSIZE+=1;
	
	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	printf("(XSIZE,YSIZE)::(%d,%d)\n",XSIZE,YSIZE);

	datTxt = fopen(argv[1],"r");
	if(datTxt == NULL) {
		//printf("Cannot open file: %s\nCheck if file exists\n",argv[1]);
		exit(1);
	}
//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//

	printf("(XSIZE,YSIZE):(%d,%d)\n",XSIZE,YSIZE);
	//printf("Maximum size possible = %f\nTotal size of current data(XSIZE * YSIZE) = %zd\n",MAX_XSIZE_POSSIBLE,XSIZE * YSIZE);
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");




//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//


	int* data;

	data = (int*)malloc(YSIZE * XSIZE * sizeof(int));

	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[XSIZE * 10 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	int Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 7 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	printf("Working1\n");
	while(fgets(line,XSIZE *10 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		//printf("Working2\n");
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,' ');
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atoi(tempVal);
				*(data + j * XSIZE + i) = Value;
				fprintf(inpCheck,"%d ",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atoi(tempVal);
				*(data + j * XSIZE + i) = Value;
				fprintf(inpCheck,"%d\n",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);
			}
		}
		
		j++;
	}

	fclose(datTxt);
	fclose(datTxt1);
	fclose(inpCheck);

	printf("Done data[%zd][%zd] = %d\n",j-1,i-1,*(data + 500 * XSIZE + 500));	
	printf("Working File IO\n");


//-----------Getting the number of devices and their sizes------------------------------------------------//
        int DeviceCount;
        DeviceCount = Get_GPU_devices();


//------------------Initializing the structures that will hold GPU data-----------------------------------//
	GPU_struct GPU_values[DeviceCount];
	//HANDLE_ERROR(hipHostMalloc((void**)&GPU_values,DeviceCount*sizeof GPU_values));
//--------------------------------------------------------------------------------------------------------//

	//XSIZE  = number of total columns
	//YSIZE = number of total rows

	//Variable that holds YSIZE initially. This changes as number of rows 
	//for each GPU is calculated
	int tmpSize = 0;
	//Variable needed to compute the total rows each GPU will have
	int count = 0;
	//offset holds either 2*RADIUS or RADIUS depending on the part of data
	int offset = 0;
	//sum of the total positions of the rows for each GPU in each iteration
	int pos = 0;
	//Actual position inside the data matrix
	size_t data_position = 0;

	tmpSize = YSIZE;
	count = DeviceCount;



	//Iterating through all the available devices
	for(i = 0;i<DeviceCount;i++){
		printf("\n########################Device %d #############################\n",i);

		//If the total rows are not exactly divisible by the number of GPUs; add 1
		if(tmpSize % count != 0){
			GPU_values[i].NumRows = (tmpSize/count) + 1;
			GPU_values[i].NumCols = XSIZE;
		}else{
			GPU_values[i].NumRows = tmpSize/count;
			GPU_values[i].NumCols = XSIZE;
		}	
		//Values change here as the num of rows for each gpu is 
		//calculated after each iteration
		tmpSize = tmpSize - GPU_values[i].NumRows;
		count--;
		printf("Row Value is: %d\n",GPU_values[i].NumRows);

		if((i == 0) ||(i == (DeviceCount -1))){
			GPU_values[i].size = (GPU_values[i].NumRows + RADIUS ) * XSIZE;	
			printf("i is: %d\n",i);
		//Sections in between
		}else{
			GPU_values[i].size = (GPU_values[i].NumRows + 2*RADIUS) * XSIZE;
			//offset = RADIUS * -1;
		}
		printf("Size is: GPU_values[%zd].NumRows + RADIUS = (%d + %d )*%d *%d =  %ld\n",i,GPU_values[i].NumRows,RADIUS,XSIZE,sizeof(float),GPU_values[i].size);	
	}





	for(i = 0;i<DeviceCount;i++){

		printf("\n########################Device %d #############################\n",i);

		//-----------------Matrix Allocations----------------------------//
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipStreamCreate(&GPU_values[i].stream));

		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_data,GPU_values[i].size *sizeof(int)));	
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_angle,ANGLESIZE * sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));

		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_data,GPU_values[i].size * sizeof(int)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_angle,ANGLESIZE * sizeof(float)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		//---------------Initialization of data arrays for each GPU---------// 

		for(j=0;j<ANGLESIZE;j++) {
			GPU_values[i].h_angle[j] = j * 5 * PI/180;	
		}


		data_position = (pos + offset ) * XSIZE;

		printf("pos = %d,data_position after sub Index = %zd\n",pos,data_position);
		//Initializing the data arrays in each of the gpu with portions of the main data
		for(j=0;j<GPU_values[i].size;j++){
			GPU_values[i].h_data[j] = *(data + data_position+j);
			
			//if(j!=0 && j % 501 == 0) printf("\n");
			//printf("%d ",GPU_values[i].h_data[j]);
		}
		
		printf("Data array assigned \n");
		offset = RADIUS * -1;
		pos+=GPU_values[i].NumRows;	
	}


	for(i=0;i<DeviceCount;i++){
		
		HANDLE_ERROR(hipSetDevice(i));

		//-----------------Sending data to GPU----------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_data,GPU_values[i].h_data,GPU_values[i].size * sizeof(int),hipMemcpyHostToDevice,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_angle,GPU_values[i].h_angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice,GPU_values[i].stream));


		//----------------Kernel Variables---------------------//
		dim3 gridSize((GPU_values[i].NumCols + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK ,(GPU_values[i].NumRows+RADIUS),1);
		dim3 blockSize(THREADS_PER_BLOCK,1,1);

		//----------------Launching the Kernel---------------------//
		getMatrix<<<gridSize,blockSize,0,GPU_values[i].stream>>>(GPU_values[i].d_data,GPU_values[i].d_angle,GPU_values[i].d_anisotropy,GPU_values[i].d_azimuth,GPU_values[i].NumCols,GPU_values[i].NumRows);

		//HANDLE_ERROR(hipDeviceSynchronize());

		//---------------Getting data back------------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_anisotropy,GPU_values[i].d_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_azimuth,GPU_values[i].d_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost,GPU_values[i].stream));
	}



	for(i=0;i<DeviceCount;i++){
		//------------------------------------Freeing data---------------------------------------//
		HANDLE_ERROR(hipSetDevice(i));	
		HANDLE_ERROR(hipStreamSynchronize(GPU_values[i].stream));

		HANDLE_ERROR(hipFree(GPU_values[i].d_anisotropy));
		HANDLE_ERROR(hipFree(GPU_values[i].d_azimuth));
		HANDLE_ERROR(hipFree(GPU_values[i].d_data));
		HANDLE_ERROR(hipFree(GPU_values[i].d_angle));

		HANDLE_ERROR(hipHostFree(GPU_values[i].h_data));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_angle));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_anisotropy));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_azimuth));

		HANDLE_ERROR(hipStreamDestroy(GPU_values[i].stream));

			
	}

	/*for(i=0;i<DeviceCount;i++){
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_data));
		hipDeviceReset();
	}*/

	
	
	free(data);
	
	return 0;
}
		
