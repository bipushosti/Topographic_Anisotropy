#include "hip/hip_runtime.h"
//Input file: space delimited

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>

#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
#include <assert.h>
#include "topographic_anisotropy_largerGrid.h"

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Size of the GPU memory
#define GPU_MEMSIZE_GB		6

//For case in which XSIZE = 1201 and YSIZE = 801
#define GLOBAL_MEM_USE_MB	773
#define MEM_USE_PER_THREAD_B	1280

//MAX_XSIZE_POSSIBLE is the maximum size of x or max number of columns if there is only one row
#define MAX_XSIZE_POSSIBLE	7483647
//#define MAX_XSIZE_POSSIBLE	floor(((GPU_MEMSIZE_GB * 1000 - GLOBAL_MEM_USE_MB)*1000000)/MEM_USE_PER_THREAD_B) 


//#define XSIZE 		1201
//#define YSIZE			801


//Always have even number of radius;and divisible by 10
#define RADIUS			100

#define	RADSTEP			1
#define ANGLESIZE		36	//Size of angle array	

#define PI 			3.14159


#define THREADS_PER_BLOCK	512

//__constant__ int RADIUS;

//#define FILENAME	"Annie_coastDEM.txt"
//---------------------------Function and Global variable declarations--------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,long int XSIZE,long int YSIZE);
int Get_GPU_devices();
static void HandleError(hipError_t err,const char *file, int line);
inline hipError_t checkCuda(hipError_t result);


//--------------------------------------------------------------------------------------------------------------------------//

__global__ void getMatrix(int* data,float* angle,float* anisotropy,float* azimuth,long int XSIZE,long int YSIZE)
{

	//The kernel does not use the new definition of RADIUS in main but the one at the top of the file
	//Therefore the define at the top and the input value of RADIUS must be equal (For now)

	//printf("The RADIUS is: %d\n",RADIUS);
	

//	Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//----------------------------------------------------------------------------------------------------------------------------//	

	//y>YSIZE - 1 for multi-gpu code because the computation needs to be done upto the radius
	if((y>(YSIZE - 1))||(y<(RADIUS))) return;
	else if((x>(XSIZE - RADIUS - 1))||(x<(RADIUS))) return;
	else
	{
		//printf("%d,%d\n",y,x);
		//Actual computation
		int xrad,yrad,xradOrtho1,yradOrtho1,xradOneEighty,yradOneEighty,valueOneEighty;
		int valueOrtho1,valueOrtho2,xradOrtho2,yradOrtho2,i,j;

	
		float variance[RADIUS];
		float orientation[RADIUS];
		float ortho[RADIUS];


		float value,sum_value,avg_value;
		float sum_valueOrtho,avg_valueOrtho;

	//	Initializing declared variables
		sum_value = 0;
		avg_value = 0;
		sum_valueOrtho = 0;
		avg_valueOrtho = 0;

	//	Iniitalize variance, ortho, and orientation arrays with max float value SGR changed i<100 to i<RADIUS
		for(i=0;i<RADIUS;i++){
			variance[i] = FLT_MAX;
			ortho[i] = FLT_MAX;
			orientation[i] = FLT_MAX;
		}
				
		//Flipped
		for(i=0;i<ANGLESIZE;i++) {
			//Initializing to 0 so that the sum is zero everytime it starts
			sum_value = 0;
			sum_valueOrtho = 0;

			for(j = 0;j<RADIUS;j+=RADSTEP) {
	
				//Computation for angle of interest
				xrad = (int)lrintf(cosf(angle[i]) * (j+1) + x);	
				yrad = (int)lrintf(sinf(angle[i]) * (j+1) + y);	

				value = data[y * XSIZE + x]  - data[yrad * XSIZE + xrad];
				value = value * value;
				
				//One eighty angle computation
				xradOneEighty = (int)lrintf(cosf(angle[i]+PI) * (j+1) + x);	
				yradOneEighty = (int)lrintf(sinf(angle[i]+PI) * (j+1) + y);	
				
				valueOneEighty = data[y * XSIZE + x] - data[yradOneEighty * XSIZE + xradOneEighty];
				valueOneEighty = valueOneEighty * valueOneEighty;

				sum_value = sum_value + value + valueOneEighty;
				avg_value = sum_value/(2*(j+1)); //the average variance from scale 1 to scale j

				//Computation for values on angle orthogonal to angle of interest
				xradOrtho1 = (int)lrintf(cosf(angle[i]+PI/2) * (j+1) + x);	
				yradOrtho1 = (int)lrintf(sinf(angle[i]+PI/2) * (j+1) + y);	
				
				valueOrtho1 = data[y * XSIZE + x]  - data[yradOrtho1 * XSIZE + xradOrtho1];
				valueOrtho1 = valueOrtho1 * valueOrtho1;

				//One eighty ortho angle computation
				xradOrtho2 = (int)lrintf(cosf(angle[i]+PI*3/2) * (j+1) + x);	
				yradOrtho2 = (int)lrintf(sinf(angle[i]+PI*3/2) * (j+1) + y);	

				valueOrtho2 = data[y * XSIZE + x]  - data[yradOrtho2 * XSIZE + xradOrtho2];
				valueOrtho2 = valueOrtho2 * valueOrtho2;

				sum_valueOrtho = sum_valueOrtho + valueOrtho1 + valueOrtho2;
				avg_valueOrtho = sum_valueOrtho/(2*j+1);

				//Fail safe to ensure there is no nan or inf when taking anisotropy ratio, later on.			
				if(avg_value == 0) {
					if((avg_valueOrtho < 1) && (avg_valueOrtho > 0)) {
						avg_value = avg_valueOrtho;
					}
					else{
						avg_value = 1;
					}
				}

				if(avg_valueOrtho == 0) {
					avg_valueOrtho = 1;
				}
				
				//Determine if the variance is minimum compared to  others at scale j, if so record it and its angle i. If not, pass it
				if(avg_value < variance[j]) {
					variance[j] = avg_value;
					orientation[j] = angle[i];
					ortho[j] = avg_valueOrtho;		
				}	
			}
		}

		for(j=0;j<RADIUS;j+=RADSTEP){	
			anisotropy[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = ortho[j]/variance[j];
			azimuth[y * XSIZE  * RADIUS/RADSTEP + x * RADIUS/RADSTEP + j] = orientation[j] * 180/PI;
		}
	}
 
}

//--------------------------------------END OF KERNEL-----------------------------------------------------------//



//--------------------------------------Handle Error()-----------------------------------------------------------//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


//--------------------------------------Get_GPU_devices()-----------------------------------------------------------//

int Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));

	if(!prop.deviceOverlap){
		printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		printf("%d Devices Found\n",DeviceCount);
	}else{
		printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	for(int i = 0;i<DeviceCount;i++){
		hipDeviceProp_t properties;
		HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", properties.name);
		printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
		printf("\n");

	}

	return DeviceCount;
}



//#################################################################################################################################//
//#################################################################################################################################//
//#################################################################################################################################//

//--------------------------------------Main()-----------------------------------------------------------//
int main(int argc,char* argv[])
//int main()
{

	char delimiter;

	//delimiter_string = "A";
	if(argc == 1){
		printf("\tNot enough arguments\n");
		printf("\t\tUsage: ./Executable DataFileName DataFileDelimiter Radius\n");
		return 0;
	}

	//In the future use optarg
	if(strcmp(argv[2],"space")==0){
		delimiter = ' ';
	}
	else if(strcmp(argv[2],"Space")==0){
		delimiter = ' ';
	}else{
		delimiter = *argv[2];
	}
	
	printf("Delimiter: %c\n",delimiter);
	//return 0;

	#undef RADIUS
	#define RADIUS atoi(argv[3])

	//RADIUS = tmp;
	printf("Radius is %d\n",RADIUS);


//-------------------------------------------------------------------------------------//
	//Setting the output buffer to 500MB
	size_t limit;
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024));
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);
	
	//Setting the heap size 
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,10 * 1000 * 1000 * 4));

//-------------------------------------------------------------------------------------//	
	//File declarations and opening them
	FILE *datTxt1,*datTxt;
	FILE *outputAnisotropy00,*outputAnisotropy09,*outputAnisotropy24,*outputAnisotropy49,*outputAnisotropy99;
	FILE *outputAzimuth00,*outputAzimuth09,*outputAzimuth24,*outputAzimuth49,*outputAzimuth99; 
	
	FILE *outputAnisotropy04,*outputAzimuth04;

	FILE * inpCheck;
	inpCheck = fopen("inpCheck.txt","w");
	if(inpCheck == NULL) {
		perror("Cannot open inpcheck.txt file");
		return (-1);
	}
	

	datTxt1 = fopen(argv[1],"r");	
	if(datTxt1 == NULL) {
		printf("Cannot open file: %s  \nCheck if file exists.\n",argv[1]);
		exit(1);
	}

//-------------------------------------------------------------------------------------//
//				Setting Up Output Filenames
//-------------------------------------------------------------------------------------//

	char *lastSlash;
	char FileName[20];
	char AniFirst[80],AniFive[80],AniTen[80],AniTwentyFive[80],AniFifty[80],AniLast[80];
	char AziFirst[80],AziFive[80],AziTen[80],AziTwentyFive[80],AziFifty[80],AziLast[80];

	memset(FileName,'\0',sizeof(FileName));

	lastSlash = strrchr(argv[1],'/');

	if(lastSlash == NULL){
		strcpy(FileName,argv[1]);
	}
	else{
		printf("Found slash at %s\n",lastSlash);
		strcpy(FileName,lastSlash+1);
	}
	printf("FileName is %s\n",FileName);

	strcpy(AniFirst,"Out_Ani_First_");
	strcpy(AniFive,"Out_Ani_Five_");
	strcpy(AniTen,"Out_Ani_Ten_");
	strcpy(AniTwentyFive,"Out_Ani_TwentyFive_");
	strcpy(AniFifty,"Out_Ani_Fifty_");
	strcpy(AniLast,"Out_Ani_Last_");

	strcat(AniFirst,FileName);
	strcat(AniFive,FileName);
	strcat(AniTen,FileName);
	strcat(AniTwentyFive,FileName);
	strcat(AniFifty,FileName);
	strcat(AniLast,FileName);



	strcpy(AziFirst,"Out_Azi_First_");
	strcpy(AziFive,"Out_Azi_Five_");
	strcpy(AziTen,"Out_Azi_Ten_");
	strcpy(AziTwentyFive,"Out_Azi_TwentyFive_");
	strcpy(AziFifty,"Out_Azi_Fifty_");
	strcpy(AziLast,"Out_Azi_Last_");

	strcat(AziFirst,FileName);
	strcat(AziFive,FileName);
	strcat(AziTen,FileName);
	strcat(AziTwentyFive,FileName);
	strcat(AziFifty,FileName);
	strcat(AziLast,FileName);

	printf("Ani First is %s\n",AniFirst);
//-------------------------------------------------------------------------------------//


	outputAnisotropy00 = fopen(AniFirst,"a");
	outputAnisotropy04 = fopen(AniFive,"a");
	outputAnisotropy09 = fopen(AniTen,"a");
	outputAnisotropy24 = fopen(AniTwentyFive,"a");
	outputAnisotropy49 = fopen(AniFifty,"a");
	outputAnisotropy99 = fopen(AniLast,"a");
	if((outputAnisotropy00 == NULL)||(outputAnisotropy09 == NULL)||(outputAnisotropy49 == NULL)||(outputAnisotropy99 == NULL)) {
		perror("Cannot open Anisotropy file");
		return (-1);
	}

	outputAzimuth00 = fopen(AziFirst,"a");
	outputAzimuth04 = fopen(AziFive,"a");
	outputAzimuth09 = fopen(AziTen,"a");
	outputAzimuth24 = fopen(AziTwentyFive,"a");
	outputAzimuth49 = fopen(AziFifty,"a");
	outputAzimuth99 = fopen(AziLast,"a");

	if((outputAzimuth00 == NULL)||(outputAzimuth09 == NULL)||(outputAzimuth49 == NULL)||(outputAzimuth99 == NULL)) {
		perror("Cannot open Azimuth file");
		return (-1);
	}

//-----------Getting total rows and columns in the data file---------------------------------------------------------------------------------------------------//

	long int XSIZE,YSIZE;
	XSIZE = 0;
	YSIZE = 0;
	long int i,j;

	//Counting number of columns(x)
	char* max_line;
	max_line = (char*)malloc(MAX_XSIZE_POSSIBLE);
	memset(max_line,'\0',sizeof(max_line));

	fgets(max_line,MAX_XSIZE_POSSIBLE,datTxt1)!=NULL; 
	while(*max_line !='\0'){
		if(*max_line == delimiter){
			XSIZE++;
		}
		max_line++;
	}
	
	XSIZE+=1;
	
	//Counting number of rows(y)
	do{
		i = fgetc(datTxt1);
		if(i == '\n') YSIZE++;
	}while(i != EOF);
	YSIZE+=1;
	
	printf("(XSIZE,YSIZE)::(%ld,%ld)\n",XSIZE,YSIZE);

	datTxt = fopen(argv[1],"r");
	if(datTxt == NULL) {
		//printf("Cannot open file: %s\nCheck if file exists\n",argv[1]);
		exit(1);
	}
//-----------------------Checking if the data size fits the memory of the GPU----------------------------------------------------------------------------------------//

	printf("(XSIZE,YSIZE):(%ld,%ld)\n",XSIZE,YSIZE);
	//printf("Maximum size possible = %f\nTotal size of current data(XSIZE * YSIZE) = %zd\n",MAX_XSIZE_POSSIBLE,XSIZE * YSIZE);
	//(MAX_XSIZE_POSSIBLE - XSIZE*YSIZE >0)? printf("There is enough memory for the computation\n"):printf("There is not enough memory and may result in incorrect results\n");




//--------------------------------------------------------------------------------------------------------------------------------------------------------------------//


	float* data;

	data = (float*)malloc(YSIZE * XSIZE * sizeof(float));

	//XSIZE ints in a row which are max of 5 digits
	//with a space in the front and the back and space
	//between each number 
	char *startPtr,*endPtr;
	char line[XSIZE * 10 +2+(XSIZE-1)];
	memset(line, '\0', sizeof(line));
	float Value;
	i = 0;
	j = 0;
	//Assuming each number in the data set has a max of 7 characters
	char tempVal[5];
	memset(tempVal,'\0',sizeof(tempVal));

	printf("Reading the data file.\n");
	while(fgets(line,XSIZE *10 + 2 + (XSIZE-1),datTxt)!=NULL) {	
		//printf("Working2\n");
		startPtr = line;	
		for(i=0;i<XSIZE;i++) {
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));		
			if(i != (XSIZE - 1)) {	
				endPtr = strchr(startPtr,delimiter);
				strncpy(tempVal,startPtr,endPtr-startPtr); 
				Value = atof(tempVal);
				*(data + j * XSIZE + i) = Value;
				fprintf(inpCheck,"%f ",Value);
				//printf("(j,i)::(%d,%d)\n",j,i);
				//printf("Column %d\n",i);

				endPtr = endPtr + 1;
				startPtr = endPtr;
			}	
			else if(i == (XSIZE - 1)){
				strcpy(tempVal,startPtr);
				Value = atof(tempVal);
				*(data + j * XSIZE + i) = Value;
				fprintf(inpCheck,"%f\n",Value);
					
			//	printf("(j,i)::(%d,%d)\n",j,i);
		
			}
		}
		
		j++;
	}
	printf("Closing the inputdata text files. \n");
	fclose(datTxt);
	fclose(datTxt1);
	fclose(inpCheck);

	printf("Done data[%zd][%zd] = %f\n",j-1,i-1,*(data + 500 * XSIZE + 500));	
	printf("Working File IO\n");


//-----------Getting the number of devices and their sizes------------------------------------------------//
        int DeviceCount;
        DeviceCount = Get_GPU_devices();


//------------------Initializing the structures that will hold GPU data-----------------------------------//
	GPU_struct GPU_values[DeviceCount];
	//HANDLE_ERROR(hipHostMalloc((void**)&GPU_values,DeviceCount*sizeof GPU_values));
//--------------------------------------------------------------------------------------------------------//

	//XSIZE  = number of total columns
	//YSIZE = number of total rows

	//Variable that holds YSIZE initially. This changes as number of rows 
	//for each GPU is calculated
	long int tmpSize = 0;
	//Variable needed to compute the total rows each GPU will have
	int count = 0;
	//offset holds either 2*RADIUS or RADIUS depending on the part of data
	int offset = 0;
	//sum of the total positions of the rows for each GPU in each iteration
	int pos = 0;
	//Actual position inside the data matrix
	size_t data_position = 0;

	tmpSize = YSIZE;
	count = DeviceCount;



	//Iterating through all the available devices
	for(i = 0;i<DeviceCount;i++){
		printf("\n########################Device %ld #############################\n",i);

		//If the total rows are not exactly divisible by the number of GPUs; add 1
		if(tmpSize % count != 0){
			GPU_values[i].NumRows = (tmpSize/count) + 1;
			GPU_values[i].NumCols = XSIZE;
		}else{
			GPU_values[i].NumRows = tmpSize/count;
			GPU_values[i].NumCols = XSIZE;
		}	
		//Values change here as the num of rows for each gpu is 
		//calculated after each iteration
		tmpSize = tmpSize - GPU_values[i].NumRows;
		count--;
		printf("Row Value is: %ld\n",GPU_values[i].NumRows);

		if((i == 0) ||(i == (DeviceCount -1))){
			GPU_values[i].size = (GPU_values[i].NumRows + RADIUS ) * XSIZE;	
			printf("Size is: %ld\n",GPU_values[i].size);
			printf("i is: %ld\n",i);
		//Sections in between
		}else{
			GPU_values[i].size = (GPU_values[i].NumRows + 2*RADIUS) * XSIZE;
			//offset = RADIUS * -1;
		}
		printf("Size is: (GPU_values[%zd].NumRows + RADIUS) * XSIZE *sizeof(int) = (%ld + %d )*%ld *%ld =  %ld\n",i,GPU_values[i].NumRows,RADIUS,XSIZE,sizeof(float),GPU_values[i].size*sizeof(float));	
	}

	//return 0;

	for(i = 0;i<DeviceCount;i++){

		printf("\n########################Device %ld #############################\n",i);
		printf("Radius is %d\n",RADIUS);
		//-----------------Matrix Allocations----------------------------//
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipStreamCreate(&GPU_values[i].stream));
		HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, (size_t)(GPU_values[i].size *sizeof(int) + ANGLESIZE * sizeof(float) + 2*GPU_values[i].size * RADIUS/RADSTEP * sizeof(float))));

		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_data,GPU_values[i].size *sizeof(int)));	
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_angle,ANGLESIZE * sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&GPU_values[i].d_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));

		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_data,GPU_values[i].size * sizeof(int)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_angle,ANGLESIZE * sizeof(float)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		HANDLE_ERROR(hipHostMalloc((void**)&GPU_values[i].h_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float)));
		//---------------Initialization of data arrays for each GPU---------// 

		for(j=0;j<ANGLESIZE;j++) {
			GPU_values[i].h_angle[j] = j * 5 * PI/180;	
		}


		data_position = (pos + offset ) * XSIZE;

		printf("pos = %d,data_position after sub Index = %zd\n",pos,data_position);
		//Initializing the data arrays in each of the gpu with portions of the main data
		for(j=0;j<GPU_values[i].size;j++){
			GPU_values[i].h_data[j] = *(data + data_position+j);
			
			//if(j!=0 && j % 501 == 0) printf("\n");
			//printf("%d ",GPU_values[i].h_data[j]);
		}
		
		printf("Data array assigned \n");
		offset = RADIUS * -1;
		pos+=GPU_values[i].NumRows;	
	}


	for(i=0;i<DeviceCount;i++){
		HANDLE_ERROR(hipSetDevice(i));

		//-----------------Sending data to GPU----------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_data,GPU_values[i].h_data,GPU_values[i].size * sizeof(int),hipMemcpyHostToDevice,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].d_angle,GPU_values[i].h_angle,ANGLESIZE * sizeof(float),hipMemcpyHostToDevice,GPU_values[i].stream));

		//----------------Kernel Variables---------------------//
		dim3 gridSize((GPU_values[i].NumCols + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK ,(GPU_values[i].NumRows+RADIUS),1);
		dim3 blockSize(THREADS_PER_BLOCK,1,1);

		printf("GridSize(X,Y) = (%ld,%ld)\n",(GPU_values[i].NumCols + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,(GPU_values[i].NumRows+RADIUS));

		//----------------Launching the Kernel---------------------//
		printf("Radius is %d\n",RADIUS);
		getMatrix<<<gridSize,blockSize,0,GPU_values[i].stream>>>(GPU_values[i].d_data,GPU_values[i].d_angle,GPU_values[i].d_anisotropy,GPU_values[i].d_azimuth,GPU_values[i].NumCols,GPU_values[i].NumRows);

		HANDLE_ERROR(hipDeviceSynchronize());
		//getLastCudaError("Kernel failed \n");

		

		//---------------Getting data back------------------------//
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_anisotropy,GPU_values[i].d_anisotropy,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost,GPU_values[i].stream));
		HANDLE_ERROR(hipMemcpyAsync(GPU_values[i].h_azimuth,GPU_values[i].d_azimuth,GPU_values[i].size * RADIUS/RADSTEP * sizeof(float),hipMemcpyDeviceToHost,GPU_values[i].stream));

		printf("Device # %ld\n",i);
		
	}
	int z;
	for(z = 0;z<DeviceCount;z++){

		HANDLE_ERROR(hipSetDevice(z));
		HANDLE_ERROR(hipStreamSynchronize(GPU_values[z].stream));

		printf("Device l%d: Rows: %ld,Cols: %ld\n",z,GPU_values[z].NumRows,GPU_values[z].NumCols);
		printf("Radius is: %d\n",RADIUS);

		for(j=0;j<GPU_values[z].NumRows ;j++) {

			for(i=0;i<GPU_values[z].NumCols ;i++) {

				if((j>(GPU_values[z].NumRows - 1))||(j<(RADIUS))) continue;
				if((i>(GPU_values[z].NumCols - RADIUS - 1))||(i<(RADIUS))) continue;

				//printf("Col:%ld,Row: %ld\n",i,j);
				if (i == (GPU_values[z].NumCols  - RADIUS - 1)) {
					fprintf(outputAnisotropy00,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAzimuth00,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAnisotropy00,"\n");
					fprintf(outputAzimuth00,"\n");

					fprintf(outputAnisotropy04,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAzimuth04,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAnisotropy04,"\n");
					fprintf(outputAzimuth04,"\n");

					fprintf(outputAnisotropy24,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAzimuth24,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAnisotropy24,"\n");
					fprintf(outputAzimuth24,"\n");

					fprintf(outputAnisotropy49,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAzimuth49,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAnisotropy49,"\n");
					fprintf(outputAzimuth49,"\n");

					fprintf(outputAnisotropy99,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAzimuth99,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAnisotropy99,"\n");
					fprintf(outputAzimuth99,"\n");

					
				}
				else {
					fprintf(outputAnisotropy00,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAzimuth00,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 0]);
					fprintf(outputAnisotropy00,"\t");
					fprintf(outputAzimuth00,"\t");


					fprintf(outputAnisotropy04,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAzimuth04,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + 5]);
					fprintf(outputAnisotropy04,"\t");
					fprintf(outputAzimuth04,"\t");

					fprintf(outputAnisotropy24,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAzimuth24,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/4 - 1]);
					fprintf(outputAnisotropy24,"\t");
					fprintf(outputAzimuth24,"\t");

					fprintf(outputAnisotropy49,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAzimuth49,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS/2 - 1]);
					fprintf(outputAnisotropy49,"\t");
					fprintf(outputAzimuth49,"\t");

					fprintf(outputAnisotropy99,"%f",GPU_values[z].h_anisotropy[j * GPU_values[z].NumCols  * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAzimuth99,"%f",GPU_values[z].h_azimuth[j * GPU_values[z].NumCols * RADIUS/RADSTEP + i * RADIUS/RADSTEP + RADIUS - 1]);
					fprintf(outputAnisotropy99,"\t");
					fprintf(outputAzimuth99,"\t");
	
				}
					
			}
		}
	}	


	for(i=0;i<DeviceCount;i++){
		//------------------------------------Freeing data---------------------------------------//
		HANDLE_ERROR(hipSetDevice(i));	
		HANDLE_ERROR(hipStreamSynchronize(GPU_values[i].stream));
		HANDLE_ERROR(hipDeviceSynchronize());

		HANDLE_ERROR(hipFree(GPU_values[i].d_anisotropy));
		HANDLE_ERROR(hipFree(GPU_values[i].d_azimuth));
		HANDLE_ERROR(hipFree(GPU_values[i].d_data));
		HANDLE_ERROR(hipFree(GPU_values[i].d_angle));

		HANDLE_ERROR(hipHostFree(GPU_values[i].h_data));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_angle));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_anisotropy));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_azimuth));

		HANDLE_ERROR(hipStreamDestroy(GPU_values[i].stream));

		hipDeviceReset();
			
	}

	/*for(i=0;i<DeviceCount;i++){
		HANDLE_ERROR(hipSetDevice(i));
		HANDLE_ERROR(hipHostFree(GPU_values[i].h_data));
		hipDeviceReset();
	}*/

	
	
	free(data);

	fclose(outputAnisotropy00);
	fclose(outputAnisotropy04);
	fclose(outputAnisotropy09);
	fclose(outputAnisotropy24);
	fclose(outputAnisotropy49);
	fclose(outputAnisotropy99);

	fclose(outputAzimuth00);
	fclose(outputAzimuth04);
	fclose(outputAzimuth09);
	fclose(outputAzimuth24);
	fclose(outputAzimuth49);
	fclose(outputAzimuth99);

	return 0;
}
		
